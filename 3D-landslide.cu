#include "hip/hip_runtime.h"
﻿


/*
_____________________________________________________
|													   |
|     MPARS (Mesh-free Particle CFD Simulator)        |
|   Copyright (C)2014 
|_____________________________________________________|



                                      ``...........``                                               
                              ``.-::///////////////////::-..``                                      
                          `.://///////////////////////////////::..`                                 
                       .-/+++++/////////////////////////////////::::-.`                             
                    `-/+++++//////////////////////////////////::::::::::-`                          
                   -/++++++////////////////////////////////::::::::::::::::-.                       
                 `/++++++///////:--....--:////////////////::::::::::::::::::::-`                    
                ./+++++/////:.`            .:////////////:::::::::::::::::::::::-.                  
               `/+++++///:.                 `/////////::::: Copyright :::::::::::-.`               
               :+++////-`                   `:///////:::::::: AHMAD ::::::::::::::-.`             
               /++///:`                     .//////::::::: SHAKIBAEINIA ::::::::::::::-.            
              `/////.                      `://///:::::::::::::::::::::::::::::::::::----`          
              `////.                      `:///:::::::::::::::::::::::::::::::::::::------.         
               ///.                      .://:::::::::::::::::::::::::::::::::::::---------.        
               ./-                     `-://::::::::::::::::::::::::::::::::::::------------.       
                .                    `-://::::::::::::::::::::::::::::-----------------------.      
                                   `-:///::::::::::::::::::::---..`````                  `````      
                                `.:://::::::::::::::::::--.``         .:/-. .://:. .-/:.  `..`      
                             `.-:////:::::::::::::--..``      `-+o+/.ohhyhyoyhyyhy+yhyyho/yyyyo.    
                         `.-::////:::::::::::--.``      ./sss/+hhyyhyyhyyyhshhyyhyohhyyhyhhyyyho    
                   ``.-::://////::::::::--.``     `-:/:-yhyyyhyhhyyho.////:`.:/::.`:+//:.:oooo+.    
           ```..--::///////////:::::-.``    `://:.ohhyhhyyyyyy/.:---` ..``. `.```. ..``.` ....`     
       `.-:::////////////:::::--.``   `.::.-yhyyhyyhyyyho.:::. ..```` .```.``.```` ..`````-`  .`    
           ````.........````    `---`-shhyhyhhyyyh/:///:`-```.``.`````://:-`.+++/. -///:` .---`     
     ./oo+:``---.`  ```  `://:`/yhyyyyhhyyhh///:/. ..``.``...``:ooo+-shhyhhohhyyhh+hhhhhyoyyyyy-    
    .yhyyyhsshhhyo:syyyo/yhyyhyyhyyyhy:+++/-`.```. .````.+ssso/hhyyhhoyhhyy/syyyyo:yhhyyoyhhyhh/    
    `shyyyyohhyyyhyhyyyhyhhyyyh//++++.`.`````.```.`:///:shhyyhyoyyys/ .::-`  ....  `-//::/sys+-     
     `-:--.`:++++-:sssss-.////- `.``.`..` `.`////.ohhyhhyoysyo- ````              `-:::--..-//`     
      -```. ..``.  `...` `.```. .`  ``.:::-.yhhyhhsyyhyy:                       ./:-.-::.`.-:y+`    
      ..``` -`  `` -`  .``.` `. .-----hhhhhysyyyys.`-:.`            ``..::-`   :/.`.sso+o+os+:-`    
     -+ooo/``....  ..... ./+//-+hhhhhhhhhyys`.--.            ```-::/:::--.-//``s.``.///::://:-`     
     hhyyyhoohhhho+yyyys/hhyyhhyhhyhho`-/:-                 o/:/-.--:/:.....y+`o-.```````````-s-    
     /syyyo-hhyyhyhhhyyhsshyyys..:+:-         `..::/:       h..../ys+/s-.../y+ `:///////:.````yo.   
       ```  -/oo/``/o++:` .--.` ```````      ++//:--/o-     h..../yo/:-..:os+. ``.:o/---:s```-yo`   
 .```````     ````.---  `/+//++++oo+++++-   +/::++---/s-    h.....--..../os:` :+:-../+/::.`.-so:    
.hyyyhyyy/` `oyyyssssd:`/y++++ssysy////+h/`o/::/yy+---:y:   h....:hs+:.....+/``+-.`````..-/os+-     
.hsoohysosy/yoosdsoo+ho./y++++sssoo//+ossoo/:::hssy:----s/  h....:h+ :/.....:s. -://+++ooo+:.       
.hsysho ysssooyyhsoo+ho.+s++++yssoosso+:/s:::::/:::/:----s/ h-...:h+  .+/++ooso-   `...``           
 ::::::  ++syyo/ hyssho.+s++++do.`     :o/::/ysoo+//s///+oh/s/oooss/     ...``                      
              `   `----``-:+++o/`      /:oooo+/      .-:---.` ``                                    


                                                            
         `:oydmNmdho:          hddddddddddddhyys+:`         
       `sNMMMMMMMMMMMNo`       NMMMMMMMMMMMMMMMMMMMh/       
      `dMMMMNyo+odMMMMMd`      NMMMMMdhhhhhhhmNMMMMMMd-     
      oNMMMm`     yMMMMMo      NMMMMM-         -sMMMMMN:    
         `..      sMMMMM+      NMMMMM-           -NMMMMN.   
               `-sMMMMMy       NMMMMM-            oMMMMMs   
             :MMMMMMNy-        NMMMMM-            .MMMMMm   
             sMMMMMMmy+`       NMMMMM-            `MMMMMM`  
             :---+dMMMMNo      NMMMMM-            .MMMMMN`  
                   sMMMMMo     NMMMMM-            :MMMMMd   
                   -MMMMMN     NMMMMM-            yMMMMM+   
     .hdmNN+       /MMMMMd     NMMMMM-           sMMMMMh    
      hMMMMN+`   `+NMMMMN:     NMMMMM/-------:/sNMMMMMy`    
      `yMMMMMMmdmMMMMMMm-      NMMMMMMMMMMMMMMMMMMMMm/      
        -yNMMMMMMMMMNh/        NMMMMMMMMMMMMMMMMNho-        
           ./+oso+/-           :::::::::::::--.`            
                                           


*/

#include <stdio.h>
#include <math.h>
#include <fstream>
#include <iostream>
#include <time.h>
#include <vector>
#define STCK_SIZE 500000000

#include <algorithm>
#include <vector>
#include <thrust/sort.h>
#include <chrono>
#define THREADS_PER_BLOCK 512


using namespace std;
using namespace std::chrono;



//_______________________ Global Variables Definition ____________________________________

int I, J, K, NUM, l;

double n0 = 0, counter[21], Rho, MEUi, MEUj, NEU;
double b[10];
double Guess[10];
double A[10][10];
double M[10][10], Minv[10][10];
double *x, *y, *z, *p, *u, *v, *w, *SFX, *SFY, *SFZ;
int *PTYPE;
double *xstar, *ystar, *zstar, *ustar, *vstar, *wstar, *pnew, *phat, *unew, *vnew, *wnew, *NEUt, *TURB1, *TURB2, *n, *nstar, *MEU, *C, *RHO, *RHOSMOOTH, *MEU_Y, *p_rheo, *p_rheo_new, *II, *Inertia;
double *Tau_xx, *Tau_yy, *Tau_zz, *Tau_xy, *Tau_xz, *Tau_yz;

int    **neighb;
int    p_count;
double lambda;                          //         MPS discretization coefficient
double Xmin;						    //         Minimum x of searching grid
double Ymin;						    //         Minimum y of searching grid
double Zmin;						    //         Minimum z of searching grid
double Xmax;						    //         Maximum x of searching grid
double Ymax;						    //         Maximum y of searching grid
double Zmax;						    //         Maximum z of searching grid
int    FP;							    //         Number of Fluid particles: in this model FP calculate in each time step
int    WP;							    //         Number of wall particles
int    GP;								//         Number of ghost particles
double DT = 0.0050;                       //         Time step size
double c0;
int MAX_NEIGHB = 1500;



//**************************************
//***  Assigning Model Parameters ******
//**************************************

//*************** basic model conditions  *****************

double DL = 0.03;						   //         Average particle distance (or particle size)
double re = 0.1;                         //		   Support area radius
double BETA = 0.93;                       //         Coefficient for determination of free surface
double relaxp = 0.5;						//         Relaxation factor for pressure correction
double relaxu = 1.0;						//         Relaxation factor for u-velocity correction
double relaxv = 1.0;						//         Relaxation factor for v-velocity correction
double COURANT = 1;                       //         Courant number
double correction = 0;                    //         Correction factor to modify problem caused by shortage of ghoast particles
double coll = 0.85*DL;                   //       Minimum particle distance to prevent collision
double CC = 0.50;                         //         Collision coefficient
double EXPANSION = 0.0000001;				//         Expansion coefficient
double MAXresi = 0.001;                   //         Maximum acceptable residual for pressure calculation
double Cs = 0.18;                         //         Smogorinsky Constant (For using in SPS-LES turbulence model
double DELTA = DL / 5.0;                    //         Expansion value of background grid cells size.
double Ncorrection = 0.99;                //         Correction of n0 value to improve the incompressibility and initial pressure fluctuation
int    SP = 0;                            //         Number of storage particles
int    TP = 264815;						//         Total number of particles
int    KTYPE = 6;                         //         Kernel type
int    DIM = 3;                           //         Dimension
int    WBC = 0;                           //         Type of wall B.C.  No-Slip:0 & -1, Slip:1
int    TURB = 0;                          //         TURB=0: NO turbulence model, TURB=1: SPS turbulence model
int    Fraction_method = 2;               //         Method of calculation of volume of fraction. 1: Linear dist across the interface, 2: smoothed value

//*************** basic flow conditions  *****************
double NEU1 = 0.000002;                   //         Kinematic Viscosity
double NEU2 = 0.000002;                    //         K inematic Viscosity
double Rho1 = 1000;						    //         Density of phase 1
double Rho2 = 2500;						//         Density of phase 2
double gx = 0.0;                          //         Gravity acceleration in x direction
double gy = 0.0;                          //         Gravity acceleration in y direction
double gz = -9.806;                          //         Gravity acceleration in z direction
double VMAX = 10;	 					//         To avoid jumping particles out of domain (If v>Vmax ---> v=0)

//*************** pressure and pressure gradient Calculation parameters *****************

int    Method = 3;						//         Fully incompressible MPS: Method=1 ; Fully incompressible M-MPS: Method=2 ; Weakly compressible: Method=3 .
int    SOLVER = 1;                        //         SOLVER=1: CGM, SOLVER=2: PCGM (Fully incompressible models)
double c01 = 22;                        //         Numerical sound speed fluid 1. (Weakly compressible model)
double c02 = 22;                        //         Numerical sound speed fluid 2. (Weakly compressible model)
int    KHcorrection = 0;                  //         Khayyer and Gotoh pressure correction(1=yes, 0=no)
int    IterMax = 100;                     //         Maximum number of iterations for pressure calculation in each time step (Fully incompressible models)
double PMAX = 400000;                       //         A limit for the value of calculated pressure to avoid program crashing
double PMIN = -2.0;                        //         Minimum pressure, to avoid a high negative pressures
double GAMA = 7.0;

//*************** inflow & outflow parameters   *****************
int    Inflow_type = 0;                   //         0: No inflow,  1: Horizontal inflow section, 2: Vertical
double Inflow_velo = 0.0;                 //        inflow velocity perpendicular to the boundary
double Inflow_start_x = 0.0;              //        x coordination of inflow start point
double Inflow_start_y = 0.0;              //        y coordination of inflow start point
double Inflow_length = 0.0;               //        Length of the inflow section
int    Inflow_par_type = 1;               //         1: phase 1 particles, 2: phase 2 particles
int    counter_inflow = 1;
double Outflow_location = 0.0;           //        Location of the outflow boundary (i.e., the x coordinates) it has been assumed that the outflow is vertical
double Threshold = 0.0;                  //        Particles whose distance to the inflow are less than the (Threshold*DL), are removed from the domain

//*************** Rheological parameters   *****************

int    Fluid2_type = 1;                   // Newtonian:0  , H-B fluid:1
double N = 1.0;							//flow behaviour (power law) index
double MEU0 = 0.03;						// consistency index
double PHI = 0.70;                           // friction angle (RAD)
double PHI_wall = 0.04;                           // friction angle (RAD)
double PHI_bed = 0.04;                           // friction angle (RAD)
double PHI_2 = 0.95;                            //  second friction angle Values are based on  Minatti &  Paris (2015)
double cohes = 0;							//cohesiveness coefficient
double visc_max = 20;                // maximum viscosity uses to avoid singularity
double dg = 0.013;                    // grain size
double I0 = 0.70;                     // I0 value in Meu9I0 rheology     Values are based on  Minatti &  Paris (2015)
double mm = 200;
int    stress_cal_method = 1;        // Method 1; viscosity is directly used in momentum equation. Method 2: first the stress tensor is calculated then it used in momentum equation
int visc_itr_num = 1;
double visc_error = 0.0, visc_ave = 0.0;
double yeild_stress;





//*************** Surface tension parameters   *****************
int    surface_method = 0;                 // method of calculation of surface tension, 0:No surface tension  1:Koshizuka's model 2: CFS model (concentration based)
double SIGMA = 0.072;                       //surface tention coefficient (N/m)
double sharpness = 0.125;                  // interface sharpness  :   surface tension is added to the particles with C[I]=0.5+-sharpness
int    Fluid_surface = 1;                  // The fluid for which the suface tension is to be applied


//*************** Time parameters  *****************
double  t, T = 4.20;                       //         Simulation time (sec)
double  DT_MAX = 0.005;                   //         Maximum size of time interval allowed
double  output_time1 = 0.05;              //         Output times
double  output_time2 = 0.1;
double	output_time3 = 0.15;
double  output_time4 = 0.2;
double  output_time5 = 0.25;
double	output_time6 = 0.3;
double	output_time7 = 0.35;
double	output_time8 = 0.4;
double	output_time9 = 0.45;
double	output_time10 = 0.5;
double  output_time11 = 0.55;
double  output_time12 = 0.6;
double	output_time13 = 0.65;
double	output_time14 = 0.7;
double  output_time15 = 0.75;
double  output_time16 = 0.8;
double  output_time17 = 0.825;
double	output_time18 = 0.85;
double  output_time19 = 0.875;
double  output_time20 = 0.9;
double	output_time21 = 0.925;
double	output_time22 = 0.95;
double	output_time23 = 0.975;
double	output_time24 = 1;
double	output_time25 = 1.025;
double  output_time26 = 1.05;
double  output_time27 = 1.075;
double  output_time28 = 1.1;
double  output_time29 = 1.125;
double	output_time30 = 1.15;
double  output_time31 = 1.175;
double  output_time32 = 1.2;
double  output_time33 = 1.225;
double	output_time34 = 1.25;
double  output_time35 = 1.275;
double  output_time36 = 1.3;
double  output_time37 = 1.325;
double	output_time38 = 1.35;
double  output_time39 = 1.375;
double  output_time40 = 1.4;


double tec_out_intrval = 0.05;            // time interval for tecplot output




//_________________________________________________________________________________________________
// ######   ##    ##  ######   #######   ##    ## ######### ########  ###   ##  #######  ######
// ######   ##    ##  ######   #######   ##    ## ######### ########  ###   ##  #######  ######
//##    ##  ##    ##  ##   ##  ##    ##  ##    ##    ##        ##     ###   ##  ##      ##    ##
//##    ##  ##    ##  ##   ##  ##    ##  ##    ##    ##        ##     ###   ##  ##      ##    ##
//##        ##    ##  ##   ##  ##    ##  ##    ##    ##        ##     ## #  ##  ##      ##
// #####    ##    ##  ######   ##   ##   ##    ##    ##        ##     ## ## ##  ##       #####
// #####    ##    ##  ######   ##   ##   ##    ##    ##        ##     ## ## ##  #######  #####
//   #####  ##    ##  ##   ##  ######    ##    ##    ##        ##     ##  # ##  #######    #####
//   #####  ##    ##  ##   ##  ######    ##    ##    ##        ##     ##  # ##  ##          #####
//       ## ##    ##  ##    ## ##   ##   ##    ##    ##        ##     ##   ###  ##              ##
// ##    ## ##    ##  ##    ## ##    ##  ##    ##    ##        ##     ##    ##  ##        ##    ##
// ##    ## ##    ##  ##    ## ##    ##  ##    ##    ##        ##     ##    ##  ##        ##    ##
//  ######   ######   #######  ##    ##   ######     ##     ########  ##    ##  #######   ######
//  ######   ######   #######  ##    ##   ######     ##     ########  ##    ##  #######   ######


//===========================================================================================
//=====================     Kernel function       ===========================================
//===========================================================================================

//----------- This subroutine generate weight function or kernel function  --------- ----------
//-- Input variables are particle distance (R), kernel type, Space imension and support radius-----------
//            ________________________________________________
//           |   No. |    Kernel type                         |
//            ================================================
//           |...1...|Second order polynomial function        |
//           |...2...|Proportioal function (Kushizuka, 1998)  |
//           |...3...|Cubic splin func.                       |
//           |...4...|Quartic function                        |
//           |...5...|Quartic splin func.                     |
//           |___6___|________________________________________|

double W(double R, int KTYPE, int dim)
{
	double w;
	double q = R / re;
	//------------------------------------
	if (KTYPE == 1)                      // Second order polynomialfunction (Koshizuka and Oka 1996)
	{
		if (q<0.5)         w = 2.0 - 4.0*pow(q, 2);
		else if (q <= 1.0)   w = (2 * q - 2)*(2 * q - 2);
		else               w = 0;
	}
	//------------------------------------
	if (KTYPE == 2)                    // Rational function (Koshizuka and Oka 1998)
	{
		if (q <= 1.0)        w = (1 / q - 1);
		else               w = 0;
	}
	//------------------------------------
	if (KTYPE == 3)
	{
		double C;
		if (dim == 1) C = 0.6666;
		if (dim == 2) C = 1.43*3.14;
		if (dim == 3) C = 1 / 3.14;
		if (R<re)                     w = C / pow(re, dim)*(1 - 1.5*pow(q, 2.0) + 0.75*pow(q, 3.0));
		else if (R >= re && R<(2.0*re)) w = C / pow(re, dim)*(0.25*pow(2.0 - q, 3.0));
		else                          w = 0;
	}
	//------------------------------------

	if (KTYPE == 5)
	{

		if (q <= 1.0)                w = 1.5*log(1 / (q + 0.000001));
		else                       w = 0;
	}
	//--------------------------------

	if (KTYPE == 6)           // #rd oerder polynomial function (Shakibaeinia and Jin 2010)
	{

		if (q <= 1.0)                w = pow((1 - q), 3);
		else                       w = 0;

	}


	return (w);
}
//______________________________  END OF SUBROUTINE (W)  ____________________________________________


//=================================================================================================
//=====================     XIJ Calculation    ====================================================
//=================================================================================================
double DX(int i, int j)
{
	return (x[j] - x[i]);
}
//______________________________  END OF SUBROUTINE (DX)  __________________________________________

//================================================================================================
//=====================     XIJ* Calculation    ==================================================
//================================================================================================
double DXSTAR(int i, int j)
{
	return (xstar[j] - xstar[i]);
}
//______________________________  END OF SUBROUTINE (DXSTAR)  _______________________________________

//=================================================================================================
//=====================     YIJ Calculation     ===================================================
//=================================================================================================
double DY(int i, int j)
{
	return (y[j] - y[i]);
}
//______________________________  END OF SUBROUTINE (DY)  ___________________________________________


//=================================================================================================
//=====================     YIJ* Calculation     ==================================================
//=================================================================================================
double DYSTAR(int i, int j)
{
	return (ystar[j] - ystar[i]);
}
//______________________________  END OF SUBROUTINE (DYSTAR)  _______________________________________



//=================================================================================================
//=====================     ZIJ Calculation     ===================================================
//=================================================================================================
double DZ(int i, int j)
{
	return (z[j] - z[i]);
}
//______________________________  END OF SUBROUTINE (DZ)  ___________________________________________


//=================================================================================================
//=====================     ZIJ* Calculation     ==================================================
//=================================================================================================
double DZSTAR(int i, int j)
{
	return (zstar[j] - zstar[i]);
}
//______________________________  END OF SUBROUTINE (DZSTAR)  _______________________________________


//=================================================================================================
//=====================     Distance Calculation     ==============================================
//=================================================================================================
double DIST(int i, int j)
{
	double R;
	R = sqrt(pow(DX(i, j), 2.0) + pow(DY(i, j), 2.0) + pow(DZ(i, j), 2.0));
	return (R);
}
//______________________________  END OF SUBROUTINE (DIST)  _________________________________________


//=================================================================================================
//=====================     Distance Calculation     ==============================================
//=================================================================================================
double DISTSTAR(int i, int j)
{
	double R;
	R = sqrt(pow(DXSTAR(i, j), 2.0) + pow(DYSTAR(i, j), 2.0) + pow(DZSTAR(i, j), 2.0));
	return (R);
}
//______________________________  END OF SUBROUTINE (DISTSTAR)  _____________________________________



//=================================================================================================
//=====================     Particle Number Density Calc.  ========================================
//=================================================================================================
double PNUM(int I, int ktype, int dim, int num)
{
	double sum = 0.0;
	double d = 0;
	for (l = 2; l <= neighb[I][1]; l++)
	{
		J = neighb[I][l];
		d = DIST(I, J);
		if (I != J) sum = sum + W(d, ktype, dim);
	}
	if (KTYPE != 2)sum = sum*Ncorrection;
	return(sum);
}
//______________________________  END OF SUBROUTINE PNUM  ___________________________________________


//=================================================================================================
//=====================     Particle Number Density Calc.  ========================================
//=================================================================================================
double PNUMSTAR(int I, int ktype, int dim, int num)
{
	double sum = 0.0;
	double d = 0;
	for (l = 2; l <= neighb[I][1]; l++)
	{
		J = neighb[I][l];
		d = DISTSTAR(I, J);
		if (I != J) sum = sum + W(d, ktype, dim);
	}

	return(sum);
}
//______________________________  END OF SUBROUTINE PNUMSTAR  _______________________________________



//=================================================================================================
//==============   Conjugate Gradient Method For Linear Eq.s   ====================================
//=================================================================================================
//   ______________________________________________________________________________________
//  | This SUBROUTINE solve linear symmetric posetive definite system of equation (Ax=b)   |
//  | using conjugate gradient method. This Subroutine only change initial guess matrix      |
//  | to answer matrix and does not return any value.                                      |
//  |   Notice: A,b and Guess matrixes are global and should defined out of this function. |
//  |______________________________________________________________________________________|

//     NOTE: This subroutine has not been update since the initial code development in 2007 and 2008, Therefore it may need some modification to be used in the present code

double CGM(int size)
{
	double r1[13000], r2[13000], x1[13000], x2[13000], p1[13000], p2[13000];
	double alpha, beta, sum, sum1, sum2, redidual = 10;
	int k, m;

	//---------------------------- Initiation --------------------------
	for (k = 1; k <= size; k++)
	{
		x1[k] = Guess[k];        //replasing guess of initial answer matrix
	}

	for (k = 1; k <= size; k++)
	{
		sum = 0;
		for (m = 1; m <= size; m++)
		{
			sum = sum + A[k][m] * x1[m];  // calculation of initial errors and p1
		}
		r1[k] = b[k] - sum;
		p1[k] = r1[k];
	}
	//------------------------- repeatation loop -------------------------
	for (int repeat = 1; repeat<IterMax; repeat++)
	{
		sum1 = 0;
		sum2 = 0;
		for (k = 1; k <= size; k++)              //Calculation of alpha
		{
			sum = 0;
			for (m = 1; m <= size; m++)
			{
				sum = sum + A[k][m] * p1[m];
			}
			sum1 = sum1 + sum*p1[k];
			sum2 = sum2 + r1[k] * r1[k];
		}
		alpha = sum2 / sum1;

		for (k = 1; k <= size; k++)         //Calculation of new answers & errors
		{

			sum = 0;
			for (m = 1; m <= size; m++)
			{
				sum = sum + A[k][m] * p1[m];
			}
			r2[k] = r1[k] - alpha*sum;
			x2[k] = x1[k] + alpha*p1[k];
		}

		sum1 = 0;
		sum2 = 0;
		for (k = 1; k <= size; k++)         //Calculation of Beta
		{
			sum1 = sum1 + r1[k] * r1[k];
			sum2 = sum2 + r2[k] * r2[k];
		}
		beta = sum2 / sum1;

		sum = 0;
		for (k = 1; k <= size; k++)        //replacement of values for new repeat
		{
			p2[k] = r2[k] + beta*p1[k];
			p1[k] = p2[k];
			r1[k] = r2[k];
			x1[k] = x2[k];
			sum = sum + fabs(r2[k]);
		}
		redidual = sum / size;                      //calculation of average error to stop the loop
		if (redidual<MAXresi)repeat = IterMax;    // End of calculation if residual is less than maximum acceptable residual
	}

	for (k = 1; k <= size; k++)        //replacement of values for new reoeat
	{
		Guess[k] = x1[k];
	}
	return (0);
}
//______________________________  END OF SUBROUTINE (CGM) ___________________________________________



//=================================================================================================
//=================  preparing MATRIXES (A,b and guess for x) for CGM  ============================
//=================================================================================================

//     NOTE: This subroutine has not been update since the initial code development in 2007 and 2008, Therefore it may need some modification to be used in the present code
double MATRIX()
{
	int K = 0;
	double sum = 0, sum1 = 0, d = 0;
	//----- preparing matrixes (A,b and guess for x) for CGM by Original MPS-------
	if (Method == 1)
	{
		for (I = 1; I <= (FP + WP); I++)
		{
			sum1 = 0;
			for (J = 1; J <= NUM; J++)
			{
				d = DIST(I, J);
				if (I != J && d>coll / 10 && d<2 * re) sum1 = sum1 + W(d, KTYPE, 2);
			}


			if (PTYPE[I] == 0) correction = 1.0; else correction = 1.0;
			if (nstar[I]<BETA*n0)
			{
				b[I] = 0.00;
			}
			else
			{
				sum = 0;
				for (J = 1 + FP + WP; J <= NUM; J++)
				{
					d = DIST(I, J);
					if (d<(2.0*re) && d>coll / 10)sum = sum + pnew[J] * W(d, KTYPE, 2);
				}
				b[I] = correction*(-Rho*relaxp*lambda / (4 * pow(DT, 2)))*(nstar[I] - n0) - sum;

			}
			Guess[I] = p[I];
			for (J = 1; J <= (FP + WP); J++)
			{
				d = DIST(I, J);
				if (I == J)    { A[I][J] = -n[I] * correction; }

				//		if (I==J)A[I][J]=-sum1-EXPANSION*Rho*lambda*n0/(4*pow(DT,2));

				else if (d<(2.0*re) && d>coll / 10) A[I][J] = W(d, KTYPE, 2);
				else A[I][J] = 0;
			}
		}
	}



	//----- preparing matrixes (A,b and guess for x) for CGM by Modified MPS-------
	if (Method == 2)
	{
		for (I = 1; I <= NUM - GP; I++)
		{
			if (nstar[I]<BETA*n0)
			{
				b[I] = 0.00;
			}
			else
			{
				sum = 0;
				for (J = 3088; J <= NUM; J++)
				{
					sum = sum + pnew[J] * W(DIST(I, J), KTYPE, 2);
				}
				b[I] = (-Rho / (4 * pow(DT, 2)))*(nstar[I] - n0) - sum;
			}

			Guess[I] = p[I];
			for (J = 1; J <= NUM - GP; J++)
			{
				if (I == J)
				{
					sum = 0;
					for (K = 1; K <= NUM; K++)
					{
						if (I != K && DIST(I, K)<(2 * re) && DIST(I, K)>0.001)
							sum = sum + ((1.0 / DIST(I, K))*W(DIST(I, K), KTYPE, 2));
					}
					A[I][J] = -sum;
				}
				else if (DIST(I, J)<(2.0*re)) A[I][J] = W(DIST(I, J), KTYPE, 2) / DIST(I, J);
				else A[I][J] = 0;

			}
		}
	}
	//-------------------------------------------------------------------------------


	if (Method == 3)
	{

	}
	//-------------------------------------------------------------------------------
	return(0);
}
//______________________________  END OF Subroutine (MATRIX) ________________________________________




//==================================================================================================
//========================        Boundary condition (BC)      =====================================
//==================================================================================================

//----This subroutine assigne the values to boundariy particles (i.e. ghost and wall particles)

double BC(int slip)
{
	double MINIMUM = 100;
	int k1 = TP + 1, k2 = TP + 1;

	//-----------------------------------------------------

	if (PTYPE[I] == -2)                 // bottom
	{
		for (l = 2; l <= neighb[I][1]; l++)
		{
			J = neighb[I][l];
			/*	if ((PTYPE[J]>0) && z[J] <= 0.0 + DL)
			{
			MINIMUM = 100;
			if (fabs(DX(I, J))<MINIMUM)
			{
			k1 = J;
			MINIMUM = fabs(DX(I, J));
			}
			}*/

			if (PTYPE[J] == 0 && x[J] <= x[I] + DL / 2.0 && x[J] >= x[I] - DL / 2.0 && z[J]<0.0 + DL)
			{
				k2 = J;
			}

		}
		w[I] = 0.0, wstar[I] = 0.0, wnew[I] = 0.0;
		u[I] = 0.0, ustar[I] = 0.0, unew[I] = 0.0;
		v[I] = 0.0, vstar[I] = 0.0, vnew[I] = 0.0;

		p[I] = p[k2], pnew[I] = pnew[k2];
	}
	//-----------------------------------------------------
	//-----------------------------------------------------

	else if (PTYPE[I] == -1)                 // right walls
	{
		for (l = 2; l <= neighb[I][1]; l++)
		{
			J = neighb[I][l];
			/*	if ((PTYPE[J]>0) && x[J] >= 2.52 - DL)
			{
			MINIMUM = 100;
			if (fabs(DY(I, J))<MINIMUM)
			{
			k1 = J;
			MINIMUM = fabs(DY(I, J));
			}
			}*/
			if (PTYPE[J] == 0 && z[J] <= z[I] + DL / 2 && z[J] >= z[I] - DL / 2 && x[J] >= 2.52 - DL)
			{
				k2 = J;
			}

		}

		w[I] = 0.0, wstar[I] = 0.0, wnew[I] = 0.0;
		u[I] = 0.0, ustar[I] = 0.0, unew[I] = 0.0;
		v[I] = 0.0, vstar[I] = 0.0, vnew[I] = 0.0;

		p[I] = p[k2], pnew[I] = pnew[k2];
	}
	//-----------------------------------------------------

	/*else if (PTYPE[I]==-3  )                 // left walls
	{
	for (l=2;l<=neighb[I][1];l++)
	{
	J=neighb[I][l];
	if ((PTYPE[J]>0) && x[J]<=0.0+DL)
	{
	MINIMUM=100;
	if (fabs(DY(I,J))<MINIMUM)
	{
	k1=J;
	MINIMUM=fabs(DY(I,J));
	}
	}
	if (PTYPE[J]==0  && y[J]<=y[I]+DL/2 && y[J]>=y[I]-DL/2 && x[J]<=0.0+DL)
	{
	k2=J;
	}

	}

	v[I]=slip*v[k1],vstar[I]=slip*vstar[k1],vnew[I]=slip*vnew[k1];
	u[I]=0.0,ustar[I]=0.0,unew[I]=0.0;
	p[I]=p[k2],pnew[I]=pnew[k2];
	}*/
	//-----------------------------------------------------
	else if (PTYPE[I] == 0)       // wall particles
	{
		u[I] = 0, ustar[I] = 0, unew[I] = 0;
		v[I] = 0, vstar[I] = 0, vnew[I] = 0;

	}


	return(0.0);
}
//______________________________  END OF SUBROTINE (BC) ___________________________________________



//=================================================================================================
//===============  Finding particles in viciniy of given particle  ================================
//=================================================================================================
int NEIGHBOR1(int i, int num)
{
	double min = 100.00;
	int neighbor = 13001;
	for (int j = 1; j <= num; j++)
	{
		if (DIST(i, j)<min && PTYPE[j] == 0)
		{
			min = DIST(i, j);
			neighbor = j;
		}
	}
	return (neighbor);
}
//______________________________  END OF FUNCTION (NEIGHBOR1) _____________________________________

__global__ void calcHash(double *d_x, double *d_y, double *d_z, int *d_particleHash,\
  int *d_NUM, double *d_Xmax, double *d_Xmin, double *d_re, double *d_DELTA, double *d_Ymin, \
  double *d_Ymax, double *d_Zmax, double *d_Zmin, int *d_particleid, int *d_tnc, int *ncx, int *ncy,\
  int *ncz){

  int k = threadIdx.x + blockIdx.x * blockDim.x;
  if(k < *d_NUM){


  *ncx = int((*d_Xmax - *d_Xmin) / (*d_re + *d_DELTA)) + 1;     // Number of cells in x direction
  *ncy = int((*d_Ymax - *d_Ymin) / (*d_re + *d_DELTA)) + 1;     // Number of cells in y direction
  *ncz = int((*d_Zmax - *d_Zmin) / (*d_re + *d_DELTA)) + 1;     // Number of cells in z direction
  *d_tnc = *ncx * *ncy * *ncz;

  
  int *icell, *jcell, *kcell, *cellNum;

  int sizeint = sizeof(int);
  icell = (int *)malloc(sizeint);
  jcell = (int *)malloc(sizeint);
  kcell = (int *)malloc(sizeint);
  cellNum = (int *)malloc(sizeint);
  
  *icell = int((d_x[k] - *d_Xmin) / (*d_re + *d_DELTA)) + 1;
  *jcell = int((d_y[k] - *d_Ymin) / (*d_re + *d_DELTA)) + 1;
  *kcell = int((d_z[k] - *d_Zmin) / (*d_re + *d_DELTA)) + 1;

  *cellNum = *icell + (*jcell - 1)* *ncx + (*kcell - 1)* *ncx * *ncy;

  d_particleHash[k] = *cellNum;
  d_particleid[k] = k;

  
  free(icell);
  free(jcell);
  free(kcell);
  free(cellNum);
}

}

__global__ void findCellStart(int *particleHash, int *cellStart, int *cellEnd, int *NUM){

  int k = threadIdx.x + blockIdx.x * blockDim.x; // here index value is equal to the cell number which starts with 1 
  if(k < *NUM){
  if (particleHash[k] != particleHash[k+1] and k!= *NUM - 1){
    cellEnd[particleHash[k] - 1] = k;
    cellStart[particleHash[k+1] - 1] = k+1;
  }
  if(k == *NUM - 1){
    cellEnd[particleHash[k] - 1] = k;
  }
    }

  free(&k);            
}

__global__ void createNeighbourArraysCUDA(int *d_neighb, int *cellStart, int *cellEnd, int *particleHash, int *particleid, int *ncx, int *ncy, int *ncz, int *d_max_neighb,  int *d_NUM){

  int index = threadIdx.x + blockIdx.x * blockDim.x; 

  if(index < *d_NUM){
  int pid, icell, jcell, kcell, cellNum;

  cellNum = particleHash[index]; 
  pid = particleid[index];
  
  int neighb_index = pid * (*d_max_neighb + 1);

  kcell = (cellNum - 1)/((*ncx) * (*ncy)) + 1;
  jcell = ((cellNum - 1) - ((kcell - 1)* (*ncx) * (*ncy)))/ *ncx + 1;
  icell = cellNum - 1 - *ncx * (jcell - 1) - (*ncx * *ncy)*(kcell - 1) + 1;

  int Cnum, J;
  int curr_neighb_num = 0;
  
  int row, colu, elev, m1, m2, m3, m4, m5, m6;
  if (icell == 1)m1 = 0; else m1 = -1;
  if (icell == *ncx)m2 = 0; else m2 = +1;
  if (jcell == 1)m3 = 0; else m3 = -1;
  if (jcell == *ncy)m4 = 0; else m4 = +1;
  if (kcell == 1)m5 = 0; else m5 = -1;
  if (kcell == *ncz)m6 = 0; else m6 = +1;

  for (row = m1; row <= m2; row++)
  {
    for (colu = m3; colu <= m4; colu++) 
    {
      for (elev = m5; elev <= m6; elev++)
      {

        Cnum = icell + row + (jcell - 1 + colu)* *ncx + (kcell - 1 + elev)* *ncx* *ncy;

        if (cellEnd[Cnum - 1] != -1){

        for (int JJ = cellStart[Cnum -1]; JJ <= cellEnd[Cnum - 1]; JJ++)
        {
          J = particleid[JJ];
          curr_neighb_num++;
          d_neighb[neighb_index + curr_neighb_num] = J+1; //here the index is shifted by one unit to conform to the original MPS convention
          
        }
      }
      }
    }
  }
  
  
  d_neighb[neighb_index] = curr_neighb_num;
 }
}

__global__ void InitializeCellDetails(int *cellStart, int *cellEnd, int *d_tnc){
  int index = threadIdx.x + blockIdx.x * blockDim.x; 
  if(index < *d_tnc){
  cellStart[index] = 0; cellEnd[index] = -1;
}
free(&index);
}





void neighbour_cuda_1(){

  //cout<<endl<<"Time study for neighbour_cuda_1()"<<endl;

  // ------------------ variable declarations and initializations ------------------------------

  int *d_cellEnd, *d_cellStart, *d_NUM, *d_tnc, *tnc, *d_ncx, *d_ncy, *d_ncz, *d_max_neighb;
  int *d_particleHash, *d_particleid, *d_neighb, *h_neighb, *d_sizeof_neighbours;
  double *d_x, *d_y, *d_z, *d_Xmax, *d_Xmin, *d_Ymax, *d_Ymin, *d_Zmax, *d_Zmin, *d_re, *d_DELTA;

  int arrsizeint = NUM * sizeof(int);
  int sizeint = sizeof(int);
  int arrsizedouble = NUM * sizeof(double);
  int sizedouble = sizeof(double);
  int sizeneighb = NUM * (MAX_NEIGHB + 1) * sizeof(int);
  int sizeof_neighbours = (MAX_NEIGHB + 1) * sizeof(int);

  tnc = (int *)malloc(sizeint);
  h_neighb = (int *)malloc(sizeneighb);



  hipMalloc((void **)&d_particleHash, arrsizeint);
  hipMalloc((void **)&d_particleid, arrsizeint); 
  hipMalloc((void **)&d_x, arrsizedouble);
  hipMalloc((void **)&d_y, arrsizedouble);
  hipMalloc((void **)&d_z, arrsizedouble);
  hipMalloc((void **)&d_Xmin, sizedouble);
  hipMalloc((void **)&d_Xmax, sizedouble);
  hipMalloc((void **)&d_Ymin, sizedouble);
  hipMalloc((void **)&d_Ymax, sizedouble);
  hipMalloc((void **)&d_Zmin, sizedouble);
  hipMalloc((void **)&d_Zmax, sizedouble);
  hipMalloc((void **)&d_re, sizedouble);
  hipMalloc((void **)&d_DELTA, sizedouble);
  hipMalloc((void **)&d_NUM, sizeint);
  hipMalloc((void **)&d_tnc, sizeint);
  hipMalloc((void **)&d_ncx, sizeint);
  hipMalloc((void **)&d_ncy, sizeint);
  hipMalloc((void **)&d_ncz, sizeint);
  hipMalloc((void **)&d_neighb, sizeneighb);
  hipMalloc((void **)&d_max_neighb, sizeint);
  hipMalloc((void **)&d_sizeof_neighbours, sizeof_neighbours);

  hipMemcpy(d_x, x, arrsizedouble, hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, arrsizedouble, hipMemcpyHostToDevice);
  hipMemcpy(d_z, z, arrsizedouble, hipMemcpyHostToDevice);
  hipMemcpy(d_Xmin, &Xmin, sizedouble, hipMemcpyHostToDevice);
  hipMemcpy(d_Xmax, &Xmax, sizedouble, hipMemcpyHostToDevice);
  hipMemcpy(d_Ymin, &Ymin, sizedouble, hipMemcpyHostToDevice);
  hipMemcpy(d_Ymax, &Ymax, sizedouble, hipMemcpyHostToDevice);
  hipMemcpy(d_Zmin, &Zmin, sizedouble, hipMemcpyHostToDevice);
  hipMemcpy(d_Zmax, &Zmax, sizedouble, hipMemcpyHostToDevice);
  hipMemcpy(d_re, &re, sizedouble, hipMemcpyHostToDevice);
  hipMemcpy(d_DELTA, &DELTA, sizedouble, hipMemcpyHostToDevice);
  hipMemcpy(d_NUM, &NUM, sizeint, hipMemcpyHostToDevice);
  hipMemcpy(d_max_neighb, &MAX_NEIGHB, sizeint, hipMemcpyHostToDevice);
  hipMemcpy(d_sizeof_neighbours, &sizeof_neighbours, sizeint, hipMemcpyHostToDevice);


  

  // --------------- running the calcHash kernel ----------------------------------------
 
  calcHash<<<NUM/THREADS_PER_BLOCK + 1,THREADS_PER_BLOCK>>>(d_x, d_y, d_z, d_particleHash, d_NUM, d_Xmax, d_Xmin, d_re, d_DELTA, d_Ymin, d_Ymax, d_Zmax, d_Zmin, d_particleid, d_tnc, d_ncx, d_ncy, d_ncz);
 
  // ---------------- sorting the particleHash array -----------------------------

 
  thrust::device_ptr<int> dev_Hash(d_particleHash);
  thrust::device_ptr<int> dev_id(d_particleid);
  thrust::sort_by_key(dev_Hash, dev_Hash + NUM, dev_id); //need to generalise this 10
 
  
  // --------------------- finding cell start and cell end for each cell -----------------------------

  hipMemcpy(tnc, d_tnc, sizeint, hipMemcpyDeviceToHost);
  int cellarrsize = *tnc * sizeof(int);
  hipMalloc((void **)&d_cellStart, cellarrsize); 
  hipMalloc((void **)&d_cellEnd, cellarrsize); 

 
  InitializeCellDetails<<<*tnc/THREADS_PER_BLOCK + 1,THREADS_PER_BLOCK>>>(d_cellStart, d_cellEnd, d_tnc);
  findCellStart<<<NUM/THREADS_PER_BLOCK + 1,THREADS_PER_BLOCK>>>(d_particleHash, d_cellStart, d_cellEnd, d_NUM);
 
  
  // -------------------------- Creating neighbour arrays for each particle ------------------------------

 
  createNeighbourArraysCUDA<<<NUM/THREADS_PER_BLOCK + 1,THREADS_PER_BLOCK>>>(d_neighb, d_cellStart, d_cellEnd, d_particleHash, d_particleid, d_ncx, d_ncy, d_ncz, d_max_neighb, d_NUM);

  
  
 
  hipMemcpy(h_neighb, d_neighb, sizeneighb, hipMemcpyDeviceToHost);



  // ---------------------------- Populating neighb array ----------------------
  
  
  
  for(int j=0; j<NUM; j++){
    for(int i=0; i<h_neighb[j*(MAX_NEIGHB + 1)]; i++){
      neighb[j+1][i+2] = h_neighb[j*(MAX_NEIGHB + 1) + i + 1];
    }
    neighb[j+1][1] = h_neighb[j*(MAX_NEIGHB + 1)];
  }
  
 
  
  
  // -------------------------- Deallocating memory ---------------------------

  hipFree(d_particleHash);
  hipFree(d_particleid);
  hipFree(d_cellStart);
  hipFree(d_cellEnd);
  hipFree(d_x);
  hipFree(d_y);
  hipFree(d_z);
  hipFree(d_Xmin);
  hipFree(d_Xmax);
  hipFree(d_Ymin);
  hipFree(d_Ymax);
  hipFree(d_Zmin);
  hipFree(d_Zmax);
  hipFree(d_re);
  hipFree(d_NUM);
  hipFree(d_tnc);
  hipFree(d_ncx);
  hipFree(d_ncy);
  hipFree(d_ncz);
  hipFree(d_neighb);
  hipFree(d_max_neighb);
  hipFree(d_sizeof_neighbours);

  free(h_neighb);
  free(tnc);
}

//=================================================================================================
//===============  Finding particles in viciniy of given particle  ================================
//=================================================================================================
void NEIGHBOR()
{

	// ------------------PARAMETERS DEFENTION -------------------------------------
	int ncx = int((Xmax - Xmin) / (re + DELTA)) + 1;     // Number of cells in x direction
	int ncy = int((Ymax - Ymin) / (re + DELTA)) + 1;     // Number of cells in y direction
	int ncz = int((Zmax - Zmin) / (re + DELTA)) + 1;     // Number of cells in z direction

	int tnc = ncx*ncy*ncz;							   // Total number of cells   
	int m, k, kmax, Cnum;


	int *Ista, *Iend, *nc, *icell, *jcell, *kcell;
	int *ip;                             // I is sorted number of ip[I] th paricle
	Ista = new int[tnc + 1]; //this points to the index of the first element in a cell in the array ip
	Iend = new int[tnc + 1]; //index of the last element in a cell in the array ip
	nc = new int[tnc + 1];
	icell = new int[TP + 1];
	jcell = new int[TP + 1];
	kcell = new int[TP + 1];
	ip = new int[TP + 1]; //this is the main array that we are looking for, it is sorted 
  // according to cell numbers and it contains particle indices 



	//----------------- ALLOCATING PRTICLES IN CELLS --------------------------


	for (k = 1; k <= tnc; k++) //cell loop 
	{
		Ista[k] = 1;
		Iend[k] = 0;
		nc[k] = 0;
	}

	for (k = 1; k <= NUM; k++) //particle loop
	{
		icell[k] = int((x[k] - Xmin) / (re + DELTA)) + 1;
		jcell[k] = int((y[k] - Ymin) / (re + DELTA)) + 1;
		kcell[k] = int((z[k] - Zmin) / (re + DELTA)) + 1;

		Cnum = icell[k] + (jcell[k] - 1)*ncx + (kcell[k] - 1)*ncx*ncy;     // Cell number in which particle k located

		nc[Cnum]++;						            // Number of particle in cell Cnum
		Iend[Cnum]++;						        // Number of particle in cell Cnum 

		for (m = Iend[tnc]; m >= Iend[Cnum]; m--)
		{
			if (m>0) ip[m + 1] = ip[m];
		} //this block is there to create space at the end as and when new particles are added

		for (m = Cnum + 1; m <= tnc; m++)
		{
			Ista[m]++;
			Iend[m]++;
		}

		ip[Iend[Cnum]] = k;
	}


	//--------------- FINDIND NEIGHBORS ----------------------------------
	int JJ;
	for (I = 1; I <= NUM; I++)
	{
		k = 2;
		int row, colu, elev, m1, m2, m3, m4, m5, m6;
		if (icell[I] == 1)m1 = 0; else m1 = -1;
		if (icell[I] == ncx)m2 = 0; else m2 = +1;
		if (jcell[I] == 1)m3 = 0; else m3 = -1;
		if (jcell[I] == ncy)m4 = 0; else m4 = +1;
		if (kcell[I] == 1)m5 = 0; else m5 = -1;
		if (kcell[I] == ncz)m6 = 0; else m6 = +1;

		for (row = m1; row <= m2; row++) //could be -1 to 1 , the triple loop is basically there to find all the 9 cells around that particle, including the one in which it itself is
		{
			for (colu = m3; colu <= m4; colu++) 
			{
				for (elev = m5; elev <= m6; elev++)
				{

					Cnum = icell[I] + row + (jcell[I] - 1 + colu)*ncx + (kcell[I] - 1 + elev)*ncx*ncy;

					for (JJ = Ista[Cnum]; JJ <= Iend[Cnum]; JJ++)
					{
						J = ip[JJ]; //J is tha ACTUAL particle index 
						neighb[I][k] = J;
						k++;
					}
				}
			}
		}
		kmax = k - 2;
		neighb[I][1] = kmax; //this is the total number of neighbours, which is stored at the beginning 
		//if( neighb[I][1]>1098 ||neighb[I][1]*0!=0) printf("ERROR, the neighbors of particles %d is %d", I, neighb[I][1]);
	}
	//--------------------Clearing dynamic arrays ----------------------------

	delete[]Ista;
	delete[]Iend;
	delete[]nc;
	delete[]icell;
	delete[]jcell;
	delete[]kcell;
	delete[]ip;
	Ista = NULL; Iend = NULL; nc = NULL; icell = NULL; jcell = NULL; kcell = NULL, ip = NULL;
}

//______________________________  END OF SUBROTINE (NEIGHBOR) _____________________________________


//______________________________  END OF SUBROTINE (NEIGHBOR_CUDA) _____________________________________


//================================================================================================
//====================  Collision of Particles computation =======================================
//===========================================================================================
double COLLISION(int i, double MINdistance)
{
	double cc;
	double ug, vg, wg, um, vm, wm, ur, vr, wr, vabs, d;
	double m1;
	double m2;

	if (PTYPE[i] == 1)m1 = Rho1; else m1 = Rho2;


	for (l = 2; l <= neighb[I][1]; l++)
	{
		int j = neighb[I][l];
		if (PTYPE[j] == 1) m2 = Rho1; else m2 = Rho2;

		d = DISTSTAR(i, j);
		if (i != j && d<MINdistance)
		{
			cc = CC*(MINdistance - d) / MINdistance;
			//cc=CC;
			ug = (m1*ustar[i] + m2*ustar[j]) / (m1 + m2);
			vg = (m1*vstar[i] + m2*vstar[j]) / (m1 + m2);
			wg = (m1*wstar[i] + m2*wstar[j]) / (m1 + m2);

			ur = m1*(ustar[i] - ug);
			vr = m1*(vstar[i] - vg);
			wr = m1*(wstar[i] - wg);
			vabs = (ur*DXSTAR(i, j) + vr*DYSTAR(i, j) + wr*DZSTAR(i, j)) / d;

			um = (1.0 + cc)*vabs*DXSTAR(i, j) / d;
			vm = (1.0 + cc)*vabs*DYSTAR(i, j) / d;
			wm = (1.0 + cc)*vabs*DZSTAR(i, j) / d;
			if (vabs>0)
			{
				if (PTYPE[i]>0)
				{
					ustar[i] = ustar[i] - um / m1;
					vstar[i] = vstar[i] - vm / m1;
					wstar[i] = wstar[i] - wm / m1;
					xstar[i] = xstar[i] - DT*um / m1;
					ystar[i] = ystar[i] - DT*vm / m1;
					zstar[i] = zstar[i] - DT*wm / m1;
				}

				if (PTYPE[j]>0)
				{
					ustar[j] = ustar[j] + um / m2;
					vstar[j] = vstar[j] + vm / m2;
					wstar[j] = wstar[j] + wm / m2;
					xstar[j] = xstar[j] + DT*um / m2;
					ystar[j] = ystar[j] + DT*vm / m2;
					zstar[j] = zstar[j] + DT*wm / m2;


				}
			}
		}
	}
	return (0);
}
//______________________________  END OF SUBROTINE (COLLISION) _________________________________


//===========================================================================================
//====================  Collision of Particles computation ==================================
//===========================================================================================
double COLLISION2(int i, double MINdistance)
{
	double cc;
	double ug, vg, wg, um, vm, wm, ur, vr, wr, vabs, d;
	double m1;
	double m2;

	if (PTYPE[i] == 1)m1 = Rho1; else m1 = Rho2;

	for (l = 2; l <= neighb[I][1]; l++)
	{
		int j = neighb[I][l];
		if (PTYPE[j] == 1)m2 = Rho1; else m2 = Rho2;
		d = DIST(i, j);
		if (i != j && d<MINdistance)
		{
			cc = CC*(MINdistance - d) / MINdistance;
			//cc=CC;
			ug = (m1*unew[i] + m2*unew[j]) / (m1 + m2);
			vg = (m1*vnew[i] + m2*vnew[j]) / (m1 + m2);
			wg = (m1*wnew[i] + m2*wnew[j]) / (m1 + m2);


			ur = m1*(unew[i] - ug);
			vr = m1*(vnew[i] - vg);
			wr = m1*(wnew[i] - wg);

			vabs = (ur*DX(i, j) + vr*DY(i, j) + wr*DZ(i, j)) / d;

			um = (1.0 + cc)*vabs*DX(i, j) / d;
			vm = (1.0 + cc)*vabs*DY(i, j) / d;
			wm = (1.0 + cc)*vabs*DZ(i, j) / d;


			if (vabs>0)
			{
				if (PTYPE[i]>0)
				{
					unew[i] = unew[i] - um / m1;
					vnew[i] = vnew[i] - vm / m1;
					wnew[i] = wnew[i] - wm / m1;
					x[i] = x[i] - DT*um / m1;
					y[i] = y[i] - DT*vm / m1;
					z[i] = z[i] - DT*wm / m1;
				}

				if (PTYPE[j]>0)
				{
					unew[j] = unew[j] + um / m2;
					vnew[j] = vnew[j] + vm / m2;
					wnew[j] = wnew[j] + wm / m2;

					x[j] = x[j] + DT*um / m2;
					y[j] = y[j] + DT*vm / m2;
					z[j] = z[j] + DT*wm / m2;
				}
			}
		}
	}
	return (0);
}
//______________________________  END OF SUBROTINE (COLLISION) _________________________________


//===========================================================================================
//====================  Calculation of DT to satisfy Courant no. ============================
//===========================================================================================

void DTcalculation()
{
	double max = 0;

	//	for (I=GP+WP+1; I<=NUM; I++)
	//	{
	//		if (fabs(unew[I])>max) {max=fabs(unew[I]);}
	//		if (fabs(vnew[I])>max) {max=fabs(vnew[I]);}
	//	}
	if (c01>c02)c0 = c01;
	else c0 = c02;

	max = c0;
	double courant = COURANT*DL / max;
	if (DT>courant) DT = courant;
	else if (courant<DT_MAX)DT = courant;
	else DT = DT_MAX;

	return;
}
//______________________________  END OF SUBROUTIN(DTcalculation) ____________________________



//===========================================================================================
//================ Particle reflection after collision with walls============================
//======================== Not applicable in this model======================================

void REFLECTION(double ni, double nj, double nk)
{
	// V'=V-2(V.n)n     : n:unit nornal vector

	double vn = unew[I] * ni + vnew[I] * nj + wnew[I] * nk;                                        //V.n
	unew[I] = unew[I] - 2 * ni*vn;
	vnew[I] = vnew[I] - 2 * nj*vn;
	wnew[I] = wnew[I] - 2 * nk*vn;
	return;
}
//______________________________  END OF Subroutine (REFLECTION) ____________________________


//===========================================================================================
//================================  SPS LES Turbulence Model ================================
//===========================================================================================

void SPS()
{


	double  *S12, *S11, *S22, S, d;


	double Uxx, Uxy, Uyx, Uyy;
	double w;
	int i, j;

	S12 = new double[TP + 1];
	S11 = new double[TP + 1];
	S22 = new double[TP + 1];

	for (i = GP + 1; i <= NUM; i++)
	{
		double sum1 = 0, sum2 = 0, sum3 = 0, sum4 = 0;
		for (l = 2; l <= neighb[i][1]; l++)
		{
			j = neighb[i][l];
			d = DIST(i, j);
			if (i != j && d>coll / 10)
			{
				w = W(d, KTYPE, 2);
				sum1 = sum1 + (unew[j] - unew[i])*DX(i, j)*w / d / d;
				sum2 = sum2 + (vnew[j] - vnew[i])*DY(i, j)*w / d / d;
				sum3 = sum3 + (unew[j] - unew[i])*DY(i, j)*w / d / d;
				sum4 = sum4 + (vnew[j] - vnew[i])*DX(i, j)*w / d / d;
			}
		}
		Uxx = (2.0 / n0)*sum1;
		Uyy = (2.0 / n0)*sum2;
		Uxy = (2.0 / n0)*sum3;
		Uyx = (2.0 / n0)*sum4;

		S12[i] = 0.5*(Uxy + Uyx);
		S11[i] = 0.5*(Uxx + Uxx);
		S22[i] = 0.5*(Uyy + Uyy);


		// NEUt=pow(Cs*DL,2)*(pow((Uxx+Uxx)*Uxx,0.5)+pow((Uxy+Uyx)*Uxy,0.5)+pow((Uyx+Uxy)*Uyx,0.5)+pow((Uyy+Uyy)*Uyy,0.5);
		S = 2 * (S11[i] * S11[i] + S22[i] * S22[i] + 2 * S12[i] * S12[i]);
		if (S<0) S = 0;
		S = sqrt(S);

		NEUt[i] = pow(Cs*DL, 2)*S;

		if (NEUt[i] * 0 != 0)  NEUt[i] = 0;
		if (NEUt[i]>1)     NEUt[i] = 1;


	}
	for (i = GP + 1; i <= NUM; i++)
	{
		double sum1 = 0, sum2 = 0;
		for (l = 2; l <= neighb[i][1]; l++)
		{
			j = neighb[i][l];
			d = DIST(i, j);
			if (i != j && d>coll / 10)
			{
				w = W(d, KTYPE, 2);

				sum1 = sum1 + (NEUt[j] - NEUt[i])*DX(i, j)*w / d / d;    //d(neo)/dx
				sum2 = sum2 + (NEUt[j] - NEUt[i])*DY(i, j)*w / d / d;	   //d(neo)/dy
			}
		}
		TURB1[i] = (2.0 / n0)*(2 * S11[i] * sum1 + 2 * S12[i] * sum2);       //additional terms in turbulence model (Report #8)
		TURB2[i] = (2.0 / n0)*(2 * S12[i] * sum1 + 2 * S22[i] * sum2);
	}


	delete[]S11; delete[]S12; delete[]S22;
	S11 = NULL; S12 = NULL; S22 = NULL;

	return;

}
//______________________________  END OF Subroutine (SPS) ____________________________


//===========================================================================================
//============================= Fluid particles Re-arangement================================
//===========================================================================================

void SORT()
{
	FP = 0;
	for (I = GP + WP + 1; I <= TP; I++)
		//	for (I=TP;I>=GP+WP+1;I--)
	{

		if (PTYPE[I]>0)                             // Rearange fluid particles
		{
			FP++;
			x[FP + GP + WP] = x[I];
			y[FP + GP + WP] = y[I];
			p[FP + GP + WP] = p[I];
			u[FP + GP + WP] = u[I];
			v[FP + GP + WP] = v[I];
			C[FP + GP + WP] = C[I];
			PTYPE[FP + GP + WP] = PTYPE[I];
			xstar[FP + GP + WP] = xstar[I];
			ystar[FP + GP + WP] = ystar[I];
			ustar[FP + GP + WP] = ustar[I];
			vstar[FP + GP + WP] = vstar[I];
			pnew[FP + GP + WP] = pnew[I];
			ustar[FP + GP + WP] = ustar[I];
			ustar[FP + GP + WP] = ustar[I];
			unew[FP + GP + WP] = unew[I];
			vnew[FP + GP + WP] = vnew[I];
			NEUt[FP + GP + WP] = NEUt[I];
			TURB1[FP + GP + WP] = TURB1[I];
			TURB2[FP + GP + WP] = TURB2[I];
			nstar[FP + GP + WP] = nstar[I];
		}
	}

	for (I = GP + WP + FP + 1; I <= TP; I++)                   // Rearange null particles
	{

		x[I] = 0;
		y[I] = 0;
		p[I] = 0;
		u[I] = 0;
		v[I] = 0;
		C[I] = 0;
		PTYPE[I] = -100;
		xstar[I] = 0;
		ystar[I] = 0;
		ustar[I] = 0;
		vstar[I] = 0;
		pnew[I] = 0;
		ustar[I] = 0;
		ustar[I] = 0;
		unew[I] = 0;
		vnew[I] = 0;
		NEUt[I] = 0;
		TURB1[I] = 0;
		TURB2[I] = 0;
		nstar[I] = 0;
	}

	return;
}
//______________________________  END OF Subroutine (SORT) ____________________________

//===========================================================================================
//============ Add new particle to entrance base on inflow velocity ==========================
//===========================================================================================


void INFLOW()
{

	I = FP + WP + GP + 1;
	int mm = Inflow_length / DL;
	int m = 1;

	//************* Horizontal inflow *************
	if (Inflow_type == 1)
	{
		double vv = Inflow_velo;
		double tt = DL / fabs(vv);
		if (t >= tt*counter_inflow)
		{
			for (m = 1; m <= mm; m++)
			{

				double X = Inflow_start_x + m*DL;

				x[I] = X;
				y[I] = Inflow_start_y;
				p[I] = 0;
				pnew[I] = 0;
				u[I] = 0;
				unew[I] = 0;
				v[I] = vv;
				vnew[I] = vv;
				PTYPE[I] = Inflow_par_type;
				if (PTYPE[I] == 1) RHO[I] = Rho1;
				else             RHO[I] = Rho2;

				I++;
			}
			counter_inflow++;
		}
	}
	//************* Vertical inflow *************
	if (Inflow_type == 2)
	{
		double uu = Inflow_velo;
		double tt = DL / fabs(uu);
		if (t >= tt*counter_inflow)
		{
			for (m = 1; m <= mm; m++)
			{

				double Y = Inflow_start_y + m*DL;

				x[I] = Inflow_start_x;
				y[I] = Y;
				p[I] = 0;
				pnew[I] = 0;
				u[I] = uu;
				unew[I] = uu;
				v[I] = 0.0;
				vnew[I] = 0.0;
				PTYPE[I] = Inflow_par_type;
				if (PTYPE[I] == 1) RHO[I] = Rho1;
				else             RHO[I] = Rho2;
				I++;
			}
			counter_inflow++;
		}
	}


}
//______________________________  END OF Subroutine (INFLOW) ____________________________



//===========================================================================================
//============ AVOIDING PARTICLE TO PENETRATE  BOUNDARIES  ==========================
//===========================================================================================


void BOUNDARIE()
{
	//if (x[I]<-2.00 )															{unew[I]=+fabs(unew[I]);x[I]=-2.00;}           //left wall
	if (x[I]>+2.52 - DL / 2.0)												        { unew[I] = -fabs(unew[I]); x[I] = 2.52 - DL / 2.0; }           //right wall


	if (y[I]<-1.02 + DL / 2)														  { vnew[I] = fabs(vnew[I]); y[I] = -1.02 + DL / 2.0; }            // front wall
	if (y[I]>+1.02 - DL / 2)                                                         { vnew[I] = -fabs(vnew[I]); y[I] = +1.02 - DL / 2.0; }            //back wall

	if (z[I]<0.0 + DL / 2)														      { wnew[I] = fabs(wnew[I]); z[I] = 0.0 + DL / 2; }      // bootom of the tank
	if (z[I]>Zmax)                                                                   { wnew[I] = -fabs(wnew[I]); z[I] = Zmax; }            //Top of domain





	if (x[I] < 0.5151*z[I] - 4.111)										            { unew[I] = fabs(unew[I]); x[I] = 0.5151*z[I] - 4.111 + DL / 2.0; }                     //box left wall
	if (y[I]<-0.6 && x[I] <= 0.5151*z[I] - 1.453)                                    { vnew[I] = +fabs(vnew[I]); y[I] = -0.6 + DL / 2.0; }            // box front wall
	if (y[I]> 0.6  && x[I] <= 0.5151*z[I] - 1.453)                                      { vnew[I] = -fabs(vnew[I]); y[I] = 0.6 - DL / 2.0; }            // box back wall
	if (x[I] > 0.5151*z[I] - 1.453 && t<0.25 && PTYPE[I]>1)										    { unew[I] = -fabs(unew[I]); x[I] = 0.5151*z[I] - 1.453 ; }                     //box rigth wall
	




	//double h;

	//if (t>.1)  h=0.9*(t-0.1)+0.08;
	//else h=0;

	//if (PTYPE[I]>1  && y[I]>h && x[I]>0.0605 && x[I]<0.0605+5.0*DL)      {unew[I]=-fabs(unew[I])*CC;x[I]=0.0605;}            // gate



	//if(y[I]>=Ymax-DL/2)                                   {vnew[I]=0.0;y[I]=Ymax-DL/2;}                                 //top

	double  COEFF, nx, ny, nz;

	if (x[I] >= -3.5 && x[I] <= 0.0 && z[I] <= -0.515*x[I] + DL / 2)
	{

		COEFF = 1 / sqrt(1 + 0.515*0.515);
		nx = COEFF*0.515;
		ny = 0;
		nz = COEFF;

		REFLECTION(nx, ny, nz);
		z[I] = -0.515*x[I] + DL / 2;
	}


	return;
}
//______________________________  END OF Subroutine (BOUNDARIE) ____________________________

//===========================================================================================
//============ Calculation of dynamic viscosity =================================
//===========================================================================================
void VISCOSITY(double *x_vel, double *y_vel, double *z_vel)
{
	double  *S12, *S13, *S23, *S11, *S22, *S33, d, phi = 0, phi2 = 0, grain_VF, meu0, normal_stress, *p_smooth;
	double **BL, **WL, **PS;
	int aa = 2, kx, ky;
	int kx_max = int((Xmax - Xmin) / aa / DL) + 1;
	int ky_max = int((Ymax - Ymin) / aa / DL) + 1;



	double Uxx, Uxy, Uxz, Uyx, Uyy, Uyz, Uzx, Uzy, Uzz;
	double w;

	int i, j;


	S11 = new double[TP + 1];
	S22 = new double[TP + 1];
	S33 = new double[TP + 1];
	S12 = new double[TP + 1];
	S13 = new double[TP + 1];
	S23 = new double[TP + 1];
	p_smooth = new double[TP + 1];
	BL = new double*[kx_max + 1];
	WL = new double*[kx_max + 1];
	PS = new double*[kx_max + 1];

	for (int m = 1; m <= kx_max; m++)
	{
		BL[m] = new double[ky_max + 1];
		WL[m] = new double[ky_max + 1];
		PS[m] = new double[ky_max + 1];
	}


	//-------------------------------determining the bed level
	for (kx = 1; kx <= kx_max; kx++)
	{
		for (ky = 1; ky <= ky_max; ky++)
		{
			BL[kx][ky] = Zmin;
			WL[kx][ky] = Zmin;
		}

	}

	for (i = 1; i <= NUM; i++)
	{
		kx = int((x[i] - Xmin) / aa / DL) + 1;
		ky = int((y[i] - Ymin) / aa / DL) + 1;

		if (z[i]>BL[kx][ky] && C[i]>0.5) { BL[kx][ky] = z[i]; PS[kx][ky] = pnew[i]; }
		if (z[i]>WL[kx][ky] && PTYPE[i] == 1) { WL[kx][ky] = z[i]; }
	}


	//---------------------------------------- Strain rate calculation --------------------------------------------

	for (i = 1; i <= NUM; i++)
	{

		double sum1 = 0, sum2 = 0, sum3 = 0, sum4 = 0, sum5 = 0, sum6 = 0, sum7 = 0, sum8 = 0, sum9 = 0, sum10 = 0;
		for (l = 2; l <= neighb[i][1]; l++)
		{
			j = neighb[i][l];
			d = DIST(i, j);
			if (i != j && d <= re)
			{
				w = W(d, KTYPE, 2);
				sum1 = sum1 + (x_vel[j] - x_vel[i])*DX(i, j)*w / d / d;
				sum2 = sum2 + (x_vel[j] - x_vel[i])*DY(i, j)*w / d / d;
				sum3 = sum3 + (x_vel[j] - x_vel[i])*DZ(i, j)*w / d / d;

				sum4 = sum4 + (y_vel[j] - y_vel[i])*DX(i, j)*w / d / d;
				sum5 = sum5 + (y_vel[j] - y_vel[i])*DY(i, j)*w / d / d;
				sum6 = sum6 + (y_vel[j] - y_vel[i])*DZ(i, j)*w / d / d;

				sum7 = sum7 + (z_vel[j] - z_vel[i])*DX(i, j)*w / d / d;
				sum8 = sum8 + (z_vel[j] - z_vel[i])*DY(i, j)*w / d / d;
				sum9 = sum9 + (z_vel[j] - z_vel[i])*DZ(i, j)*w / d / d;

				sum10 = sum10 + pnew[j] * w;



			}
		}
		Uxx = (DIM / n0)*sum1;
		Uxy = (DIM / n0)*sum2;
		Uxz = (DIM / n0)*sum3;

		Uyx = (DIM / n0)*sum4;
		Uyy = (DIM / n0)*sum5;
		Uyz = (DIM / n0)*sum6;

		Uzx = (DIM / n0)*sum7;
		Uzy = (DIM / n0)*sum8;
		Uzz = (DIM / n0)*sum9;






		p_smooth[i] = sum10 / n0;
		if (p_smooth[i]<0) p_smooth[i] = 0;

		S11[i] = 0.5*(Uxx + Uxx);
		S12[i] = 0.5*(Uxy + Uyx);
		S13[i] = 0.5*(Uxz + Uzx);
		S22[i] = 0.5*(Uyy + Uyy);
		S23[i] = 0.5*(Uyz + Uzy);
		S33[i] = 0.5*(Uzz + Uzz);


		//II[i] = 0.5*Uxx*Uxx + 0.5*Uyy*Uyy + 0.25*(Uxy + Uyx)*(Uxy + Uyx);
		II[i] = 0.5*(S11[i] * S11[i] + S12[i] * S12[i] + S13[i] * S13[i] + S12[i] * S12[i] + S22[i] * S22[i] + S23[i] * S23[i] + S13[i] * S13[i] + S23[i] * S23[i] + S33[i] * S33[i]);
		//	II[i]= S11[i]*S22[i] +S22[i]*S33[i]+ S11[i]*S33[i] - S12[i]*S12[i] -S13[i]*S13[i]- S23[i]*S23[i] ;
		if (II[i]<0 || II[i] * 0 != 0) II[i] = 0;

		//II=fabs(S11[i]*S22[i]-S12[i]*S12[i]);


	}




	//--------------  Newtonian visc ---------------------
	if (Fluid2_type == 0)
	{
		for (i = 1; i <= NUM; i++)
		{
			if (PTYPE[i] <= 1)MEU[i] = NEU1*Rho1;
			if (PTYPE[i] != 1)MEU[i] = NEU2*Rho2;
		}


		if (TURB>0)
		{
			NEUt[i] = Cs*DL*Cs*DL * 2 * sqrt(II[i]);

			if (NEUt[i] * 0 != 0)  NEUt[i] = 0;
			if (NEUt[i]>1)     NEUt[i] = 1;
		}
	}
	//--------------------- Granular Fluid  -------------------------
	if (Fluid2_type == 1)
	{
		for (i = 1; i <= NUM; i++)
		{


			if (TURB>0)
			{
				NEUt[i] = Cs*DL*Cs*DL * 2 * sqrt(II[i]);

				if (NEUt[i] * 0 != 0)  NEUt[i] = 0;
				if (NEUt[i]>1)     NEUt[i] = 1;
			}

			if (PTYPE[i] == 1)MEU[i] = NEU1*Rho1;
			else
			{

				phi = (C[i] - 0.25)*PHI / (1 - 0.25);
				phi2 = (C[i] - 0.25)*PHI_2 / (1 - 0.25);
				if (C[i] <= 0.25) { phi = 0.00001; phi2 = 0.00001; }
				if (PTYPE[i] <= 0) phi = PHI_bed;


				// --------------------- normal stress calculation ----------------------------
				p_rheo_new[i] = p_smooth[i];

				kx = int((x[i] - Xmin) / aa / DL) + 1;
				ky = int((y[i] - Ymin) / aa / DL) + 1;

				//		normal_stress=(BL[k]-y[i]+DL/2)*(Rho2)*9.81;                                       // normal_stress= Gama.H
				normal_stress = (BL[kx][ky] - z[i] + DL / 2)*(Rho2 - Rho1)*9.81 - (x_vel[i] * x_vel[i] + y_vel[i] * y_vel[i] + z_vel[i] * z_vel[i])*(Rho2 - Rho1) / 2.0;                                       // normal_stress= Gama.H




				if (p_smooth[i] - (WL[kx][ky] - z[i])*Rho1*9.8<0) p_smooth[i] = (WL[kx][ky] - z[i])*Rho1*9.8;
				if (t <= 1) normal_stress = 1.0*(1 - t)*(p_smooth[i] - (WL[kx][ky] - z[i])*Rho1*9.8) + 1.0*(t)*normal_stress;



				//		normal_stress=normal_stress*0.61*1500/Rho2;
				if (normal_stress < 1 || C[i] < 0.5) normal_stress = 1;

				p_rheo_new[i] = normal_stress;


				// --------------------- yeild stress calculation ----------------------------
				//	Inertia[i] = sqrt(II[i])* dg / sqrt(normal_stress / Rho2);        // Free-fall regime
				Inertia[i] = sqrt(II[i])* dg / sqrt(normal_stress / (Rho1*0.47));        // Grain inertia regime
				//	Inertia[i] = sqrt(II[i])* (NEU1*Rho1) / normal_stress ;                 //viscous regime

				grain_VF = 0.65 - (0.65 - 0.25)*Inertia[i];
				phi = phi*grain_VF / 0.65;


				yeild_stress = cohes*cos(phi) + normal_stress*sin(phi);

				if (yeild_stress < 0)yeild_stress = 0;

				visc_max = (yeild_stress*mm + MEU0);

				if (II[i]>0) MEU_Y[i] = yeild_stress*(1 - exp(-mm*sqrt(II[i]))) / 2.0 / sqrt(II[i]);
				else MEU_Y[i] = visc_max;






				// ---------------H-B rhology--------------------------------------

				//meu0 = MEU0;


				// ---------------Non-linear Meu(I) rhology--------------------------------------
				//		meu0=  0.5*    0.36               *normal_stress* dg/ (I0* sqrt(normal_stress/ Rho2)+ sqrt(II[i])*dg);               //free fall
				meu0 = 0.5*(sin(phi2) - sin(phi)) *normal_stress* dg / (I0* sqrt(normal_stress / (Rho1*0.47)) + sqrt(II[i])*dg);               //grain inertia
				//		meu0=0.5*0.36*normal_stress* (NEU1*Rho1)/ (I0* normal_stress+ sqrt(II[i])*(NEU1*Rho1));               //viscous


				// --------------linear Meu(I) rhology ----------------------------------
				//		meu0 = 0.5*(tan(phi2) - tan(phi)) * dg* sqrt(normal_stress * Rho2)      / I0;               //free fall
				//        meu0 = 0.5*(tan(phi2) - tan(phi)) * dg* sqrt(normal_stress * Rho1*0.47) / I0;               //grain inertia
				//		meu0 = 0.5*(tan(phi2) - tan(phi)) * (NEU1*Rho1)                         / I0;               //viscous



				if (II[i] <= 0 || (meu0 * 0) != 0) meu0 = MEU0;

				visc_max = (yeild_stress*mm + meu0);


				MEU[i] = MEU_Y[i] + MEU0*pow(4 * II[i], (N - 1) / 2);

				if (II[i] == 0 || MEU[i]>visc_max) MEU[i] = visc_max;
				if (PTYPE[i] <= 0) MEU[i] = MEU[i] * C[i] + 0.001*(1 - C[i]);

			}
		}

		//---------------------------------- Direct stress calculation method -----------------------------------------

		if (stress_cal_method == 2)
		{
			for (i = 1; i <= NUM; i++)
			{

				double sum1 = 0, sum2 = 0, sum3 = 0, sum4 = 0, sum5 = 0, sum6 = 0, sum7 = 0, sum8 = 0, sum9 = 0, sum10 = 0;
				for (l = 2; l <= neighb[i][1]; l++)
				{
					j = neighb[i][l];
					d = DIST(i, j);
					if (i != j && d <= re)
					{

						w = W(d, KTYPE, 2);

						double meuij = 2 * MEU[i] * MEU[j] / (MEU[i] + MEU[j]);
						if ((NEUt[i] + NEUt[j])>0) meuij = meuij + 2 * NEUt[i] * RHO[i] * NEUt[j] * RHO[j] / (NEUt[i] * RHO[i] + NEUt[j] * RHO[j]);



						sum1 = sum1 + meuij*(x_vel[j] - x_vel[i])*DX(i, j)*w / d / d;
						sum2 = sum2 + meuij*(x_vel[j] - x_vel[i])*DY(i, j)*w / d / d;
						sum3 = sum3 + meuij*(x_vel[j] - x_vel[i])*DZ(i, j)*w / d / d;

						sum4 = sum4 + meuij*(y_vel[j] - y_vel[i])*DX(i, j)*w / d / d;
						sum5 = sum5 + meuij*(y_vel[j] - y_vel[i])*DY(i, j)*w / d / d;
						sum6 = sum6 + meuij*(y_vel[j] - y_vel[i])*DZ(i, j)*w / d / d;

						sum7 = sum7 + meuij*(z_vel[j] - z_vel[i])*DX(i, j)*w / d / d;
						sum8 = sum8 + meuij*(z_vel[j] - z_vel[i])*DY(i, j)*w / d / d;
						sum9 = sum9 + meuij*(z_vel[j] - z_vel[i])*DZ(i, j)*w / d / d;

					}
				}

				Tau_xx[i] = (DIM / n0) * 2 * sum1;
				Tau_yy[i] = (DIM / n0) * 2 * sum5;
				Tau_zz[i] = (DIM / n0) * 2 * sum9;

				Tau_xy[i] = (DIM / n0)*(sum2 + sum4);
				Tau_xz[i] = (DIM / n0)*(sum3 + sum7);
				Tau_yz[i] = (DIM / n0)*(sum6 + sum8);





			}
		}



	}
	//---------------------------------------------------------------

	
	for (int m = 1; m <= kx_max; m++)
	{
		delete[] BL[m];
		delete[] WL [m];
		delete[] PS [m];
	}
	
	delete[]S11; delete[]S12; delete[]S13; delete[]S22; delete[]S23; delete[]S33; delete[]BL; delete[]WL; delete[]PS; delete[]p_smooth;




	S11 = NULL; S12 = NULL; S13 = NULL; S22 = NULL; S23 = NULL; S33 = NULL; BL = NULL; WL = NULL; PS = NULL; p_smooth = NULL;



}







//===========================================================================================
//========================== Calculation of pressure  ========================================
//===========================================================================================


void PRESSURECALC()
{

	//-------------- Using Equation of State instead of Poisson eq. -------------------

	if (Method == 3)
	{

		for (I = GP + 1; I <= NUM; I++)
		{

			if (PTYPE[I] >= 2)c0 = c02;
			else c0 = c01;




			//	Rho=Rho2*C[I]+Rho1*(1-C[I]);

			if (PTYPE[I] >= 2)Rho = Rho2;
			else Rho = Rho1;


			if (nstar[I]<BETA*n0)

			{
				pnew[I] = 0.0;

			}

			else
			{
				pnew[I] = (c0*c0*Rho / GAMA)*(pow(nstar[I] / n0, GAMA) - 1);    //P=B((rho/rho0)^7-1), B=(10*vmax)^2*Rho0/7
			}
			if (pnew[I]<PMIN)
			{
				pnew[I] = PMIN;
			}
			if (pnew[I]>PMAX)
			{
				pnew[I] = PMAX;
			}

		}

	}
	else
	{
		//---------------------- Using Conjugate gradient method -------------------

		MATRIX();
		CGM(FP + WP);
		//--------replacing result of conjugate gradient method-----------------------
		for (I = 1; I <= NUM; I++)
		{
			if (PTYPE[I]>0)
			{

				pnew[I] = Guess[GP + I];
			}

			if (nstar[I]<BETA*n0)
			{
				pnew[I] = 0.0;

			}

			if (pnew[I]<PMIN)
			{
				pnew[I] = PMIN;
			}
		}
	}


	return;
}
//______________________________  END OF Subroutine (PRESSURECALC) ____________________________

//===========================================================================================
//========================== Surface Tension model ==========================================
//===========================================================================================

//                  THIS SUBROUTINE HAS BEEN DISABLED IN THIS VESION


void Surfacetension()
{

	return;
}
//______________________________  END OF Subroutine (Surfacetension) ____________________________

//===========================================================================================
//=============== Calculation of the volume of fraction if phase II in the mixture =========
//===========================================================================================
void V_FRACTION()
{
	double sum1, sum2, d;

	if (Fraction_method == 1)   //Linear distribution
	{
		for (I = 1; I <= NUM; I++)
		{
			sum1 = 0;
			sum2 = 0;
			for (l = 2; l <= neighb[I][1]; l++)
			{
				J = neighb[I][l];

				if (I != J && PTYPE[J]>0)
				{
					sum1 = sum1 + 1;
					if (PTYPE[J] >= 2)sum2 = sum2 + 1;
				}
			}
			C[I] = sum2 / sum1;
			if (sum1 == 0)C[I] = 0;


		}
	}

	if (Fraction_method == 2)   //Non linear :  Smoothed using the weight funtion
	{
		for (I = 1; I <= NUM; I++)
		{
			sum1 = 0;
			sum2 = 0;
			for (l = 2; l <= neighb[I][1]; l++)
			{
				J = neighb[I][l];
				d = DIST(I, J);
				if (I != J && PTYPE[J]>0)
				{
					sum1 = sum1 + W(d, KTYPE, 2);
					if (PTYPE[J] >= 2)sum2 = sum2 + W(d, KTYPE, 2);
				}
			}
			C[I] = sum2 / sum1;
			if (sum1 == 0)C[I] = 0;


		}
	}

}
//______________________________  END OF FUNCTION FRACTION ____________________________




//********************************************************************************************
//           #       #    ####    ########  ###   ##
//           #       #    ####    ########  ###   ##
//           ##     ##   ##  ##      ##     ###   ##
//           ##     ##   ##  ##      ##     ###   ##
//           ###   ###  ##    ##     ##     ## #  ##
//           ###   ###  ##    ##     ##     ## #  ##
//           ## # # ##  ##    ##     ##     ## ## ##
//           ## # # ##  ##    ##     ##     ## ## ##
//           ##  #  ##  ########     ##     ##  # ##
//           ##  #  ##  ########     ##     ##  # ##
//           ##  #  ##  ##    ##     ##     ##   ###
//           ##  #  ##  ##    ##     ##     ##   ###
//           ##     ##  ##    ##  ########  ##    ##
//           ##     ##  ##    ##  ########  ##    ##
//********************************************************************************************


int main()
{
	printf(" M@@@;   s@@@@  @@@@@@@.      @@,     @@@@@@@@.    .@@@@@@   \n");
	printf(" P@@@@   @@@@@  @@@@@@@@@   .@@@@     @@@B##@@@@  #@@2sX@@@  \n");
	printf(" A@@#@, :@@@@@  @@    :@@,  @@@@@@    @@,    @@@  @@@    :@  \n");
	printf(" R@# @@ M@ @@@  @@@@@@@@@  @@@  @@@   @@@@@##@@r   @@@@@@r   \n");
	printf(" S@@ @@,@@ @@@  @@@@@@@.   @@@;;@@@   @@#9@@@5        @@@@@; \n");
	printf(" @@@ :@@@: @@@  @@        @@@@@@@@@@  @@:  .@@M   @:,    @@@ \n");
	printf(" @@@  @@@  @@@  @@       @@@     r@@, @@:    @@@. @@@@MM@@@. \n");
	printf(" @@@  :@:  @@@  @@       @@:     :@@. @@:      @@   @@@@@@   \n");

	printf("   ____________________________________________________\n");
	printf("  |          MPARS (Mesh-free Particle CFD Simulator)  |\n");
	printf("  |                        Ahmad Shakibaeinia          |\n");
	printf("  |____________________________________________________|\n");
	printf("   ____________________________________________________\n");
	printf("  |             TEST CASE: MPARS EXAMPLE 1             |\n");
	printf("  |              By: AHMAD SHAKIBAEINIA, PhD           |\n");
	printf("  |                   Date: Sept., 2015                |\n");
	printf("  |____________________________________________________|\n");
	printf("\n");



	//------------------------- Defining Dynamic Matrices -------------------------
	x = new double[TP + 1];
	y = new double[TP + 1];
	z = new double[TP + 1];
	p = new double[TP + 1];
	u = new double[TP + 1];
	v = new double[TP + 1];
	w = new double[TP + 1];
	PTYPE = new int[TP + 1];
	xstar = new double[TP + 1];
	ystar = new double[TP + 1];
	zstar = new double[TP + 1];
	ustar = new double[TP + 1];
	vstar = new double[TP + 1];
	wstar = new double[TP + 1];
	pnew = new double[TP + 1];
	phat = new double[TP + 1];
	unew = new double[TP + 1];
	vnew = new double[TP + 1];
	wnew = new double[TP + 1];
	NEUt = new double[TP + 1];
	TURB1 = new double[TP + 1];
	TURB2 = new double[TP + 1];
	p_rheo = new double[TP + 1];
	p_rheo_new = new double[TP + 1];
	C = new double[TP + 1];
	MEU = new double[TP + 1];
	RHO = new double[TP + 1];
	RHOSMOOTH = new double[TP + 1];
	SFX = new double[TP + 1];
	SFY = new double[TP + 1];
	SFZ = new double[TP + 1];
	n = new double[TP + 1];
	nstar = new double[TP + 1];
	neighb = new int*[TP + 1];
	MEU_Y = new double[TP + 1];
	II = new double[TP + 1];
	Inertia = new double[TP + 1];

	Tau_xx = new double[TP + 1];
	Tau_yy = new double[TP + 1];
	Tau_zz = new double[TP + 1];
	Tau_xy = new double[TP + 1];
	Tau_xx = new double[TP + 1];
	Tau_yz = new double[TP + 1];


	for (int m = 1; m <= TP; m++)
		neighb[m] = new int[1500];


	//---------------------- Openning INPUT and OUTPUT files ---------------------------------

	using namespace std;
	fstream in, out, history, tecplot_out;	                    // Openning  input and  output files
	fstream  out1, out2, out3, out4, out5, out6, out7, out8, out9, out10;
	fstream  out11, out12, out13, out14, out15, out16, out17, out18, out19, out20;
	fstream  out21, out22, out23, out24, out25, out26, out27, out28, out29, out30;
	fstream  out31, out32, out33, out34, out35, out36, out37, out38, out39, out40;
	in.open("input.txt", ios::in);
	out.open("output.txt", ios::out);
	history.open("history.txt", ios::out);
	tecplot_out.open("TECPLOT.txt", ios::out);


	out1.open("output1.txt", ios::out);
	out2.open("output2.txt", ios::out);
	out3.open("output3.txt", ios::out);
	out4.open("output4.txt", ios::out);
	out5.open("output5.txt", ios::out);
	out6.open("output6.txt", ios::out);
	out7.open("output7.txt", ios::out);
	out8.open("output8.txt", ios::out);
	out9.open("output9.txt", ios::out);
	out10.open("output10.txt", ios::out);
	out11.open("output11.txt", ios::out);
	out12.open("output12.txt", ios::out);
	out13.open("output13.txt", ios::out);
	out14.open("output14.txt", ios::out);
	out15.open("output15.txt", ios::out);
	out16.open("output16.txt", ios::out);
	out17.open("output17.txt", ios::out);
	out18.open("output18.txt", ios::out);
	out19.open("output19.txt", ios::out);
	out20.open("output20.txt", ios::out);
	out21.open("output21.txt", ios::out);
	out22.open("output22.txt", ios::out);
	out23.open("output23.txt", ios::out);
	out24.open("output24.txt", ios::out);
	out25.open("output25.txt", ios::out);
	out26.open("output26.txt", ios::out);
	out27.open("output27.txt", ios::out);
	out28.open("output28.txt", ios::out);
	out29.open("output29.txt", ios::out);
	out30.open("output30.txt", ios::out);
	out31.open("output31.txt", ios::out);
	out32.open("output32.txt", ios::out);
	out33.open("output33.txt", ios::out);
	out34.open("output34.txt", ios::out);
	out35.open("output35.txt", ios::out);
	out36.open("output36.txt", ios::out);
	out37.open("output37.txt", ios::out);
	out38.open("output38.txt", ios::out);
	out39.open("output39.txt", ios::out);
	out40.open("output40.txt", ios::out);



	double sum1, sum2, sum3, sum4, sum5, sum6, sum7, sum8, sum9, sum10, sum11, sum12, sum13, D, MAX = 0, weigth;

	//tecplot_out<<"VARIABLES ='x','y','ptype','u','v','p','c'\n";

	//------------------------INITIALIZATION ---------------------------------------------------
	I = 0, J = 0, K = 0, NUM = 0, l = 0;
	p_count = 1;
	int tec_counter = 0;
	for (I = 1; I <= TP + 1; I++)
	{
		x[I] = 0, y[I] = 0, z[I] = 0, p[I] = 0, u[I] = 0, v[I] = 0, w[I] = 0, PTYPE[I] = 0;
		xstar[I] = 0, ystar[I] = 0, zstar[I] = 0, ustar[I] = 0, vstar[I] = 0, wstar[I] = 0;
		pnew[I] = 0, phat[I] = 0, unew[I] = 0, vnew[I] = 0, wnew[I] = 0, NEUt[I] = 0, TURB1[I] = 0, TURB2[I] = 0, n[I] = 0, nstar[I] = 0, SFX[I] = 0, SFY[I] = 0, SFZ[I] = 0, RHO[I] = 0, C[I] = 0, RHOSMOOTH[I] = 0, MEU_Y[I] = 0, p_rheo[I] = 0, p_rheo_new[I] = 0;
	}

	//-------------------------- defining LAMDA -----------------------------------------

	if (KTYPE == 1)   lambda = 0.22143*pow(re, 2);
	if (KTYPE == 2)   lambda = 0.16832*pow(re, 2);
	if (KTYPE == 5)   lambda = 0.250*pow(re, 2);
	if (KTYPE == 6)   lambda = (3.0 / 14.0)*pow(re, 2);
	//----------------------- Reading Input file ---------------------------------------
	printf("     READING INPUT FILE . . .\n");

	Xmin = 99999999;
	Ymin = 99999999;
	Zmin = 99999999;
	Xmax = -99999999;
	Ymax = -99999999;
	Zmax = -99999999;
	FP = 0;
	WP = 0;
	GP = 0;
	TP = 0;

	in >> NUM;
	for (I = 1; I <= NUM; I++)
	{
		in >> x[I];
		in >> y[I];
		in >> z[I];
		in >> PTYPE[I];
		in >> u[I];
		in >> v[I];
		in >> w[I];
		in >> p[I];

		if (PTYPE[I]<0)  GP++;
		if (PTYPE[I] == 0) WP++;
		if (PTYPE[I]>0)  FP++;

		if (x[I]<Xmin) Xmin = x[I];
		if (x[I]>Xmax) Xmax = x[I];
		if (y[I]<Ymin) Ymin = y[I];
		if (y[I]>Ymax) Ymax = y[I];
		if (z[I]<Zmin) Zmin = z[I];
		if (z[I]>Zmax) Zmax = z[I];
	}
	TP = FP + GP + WP + SP;

	Xmin = Xmin - DL;
	Ymin = Ymin - DL;
	Zmin = Zmin - DL;

	Xmax = Xmax + DL;
	Ymax = Ymax + DL;
	Zmax = Zmax + 6 * DL;
	printf("     PRE- ITERATION CALCULATIONS... \n");
	//----------------------------------- Density assigning -----------------------------------------------------

	for (I = 1; I <= NUM; I++)
	{

		if (PTYPE[I] >= 2)RHO[I] = Rho2;
		else            RHO[I] = Rho1;

	}
	//--------------------------- Calculation of initial Particle number---------------------------------------

	neighbour_cuda_1();

	for (I = 1; I <= NUM; I++)
	{

		n[I] = PNUM(I, KTYPE, DIM, NUM);

		pnew[I] = p[I];
		if (n[I]>MAX)   MAX = n[I];
	}
	n0 = MAX;
	//printf("(%f)   ", n[144564]); getchar();

	printf("     The initial particle number density = %f\n", n0);


	//-------------------------------------------------------------------------------------
	//------------------------------- Time Iteration -------------------------------------------
	//-------------------------------------------------------------------------------------
	printf("\n");
	printf("     ITERATION STARTED  (each bar represents 1 time step) \n");
	t = 0;

	// -------------------------Calculation of CPU time	----------------------
	time_t start, end;
	double dif;
	time(&start);
	//------------------------------------------------------------
	for (int Tstep = 1; t <= T; Tstep++)
	{
		DTcalculation();
		if (((Tstep - 1) / 10.00) == int((Tstep - 1) / 10.00))
		{
			printf("     t=%f Sec     ", t);
			printf("\n");
			printf("                  ", t);
		}


		NUM = FP + WP + GP;


		if (Method != 3)
		{
			for (I = 1; I <= NUM; I++)
			{
				n[I] = PNUM(I, KTYPE, DIM, NUM);
			}
		}

		//	if (((Tstep-1)/4.00)==int((Tstep-1)/4.00))   // Setting nighboring list each 4 time step
		//	{
		neighbour_cuda_1();

		//	}
		//	if (TURB==1) SPS();
		V_FRACTION();


		//----------------------------- calculation of surface tention force --------------------------
		if (surface_method>0)Surfacetension();



		//-------------------------------------Prediction--------------------------------------

		//-------------------------------------Prediction--------------------------------------


		//----------------------------- calculation of dynamic visc --------------------------

		visc_error = 0.0, visc_ave = 0.0;

		VISCOSITY(u, v, w);

		//	 for(int visc_itr=1; visc_itr<=visc_itr_num; visc_itr++) // iteration for calculation of stress dependent viscosity
		//	 {

		for (I = 1; I <= NUM; I++)

		{
			if (PTYPE[I] >= 2)
			{
				if (C[I] > 0.5) RHO[I] = Rho2;
				else RHO[I] = C[I] * Rho2 + (1 - C[I]) * 2650;
			}

			if (PTYPE[I] <= 0)

			{

				xstar[I] = x[I];

				ystar[I] = y[I];

				zstar[I] = z[I];

				ustar[I] = u[I];

				vstar[I] = v[I];

				wstar[I] = w[I];
			}

			else

			{

				sum1 = 0.0;
				sum2 = 0.0;
				sum3 = 0.0;
				sum4 = 0.0;
				sum5 = 0.0;
				sum6 = 0.0;
				sum7 = 0.0;
				sum8 = 0.0;
				sum9 = 0.0;
				sum10 = 0.0;
				sum11 = 0.0;
				sum12 = 0.0;
				sum13 = 0.0;


				for (l = 2; l <= neighb[I][1]; l++)

				{

					J = neighb[I][l];

					D = DIST(I, J);



					if (I != J && D>0 && D <= re)

					{


						weigth = W(D, KTYPE, 2);


						if (PTYPE[I] == 1) NEU = 2 * MEU[I] * MEU[J] / (MEU[I] + MEU[J]) / Rho1;

						else NEU = 2 * MEU[I] * MEU[J] / (MEU[I] + MEU[J]) / Rho2;
						if ((NEUt[I] + NEUt[J])>0) NEU = NEU + (2 * NEUt[I] * RHO[I] * NEUt[J] * RHO[J] / (NEUt[I] * RHO[I] + NEUt[J] * RHO[J])) / RHO[I];



						sum1 = sum1 + (pnew[J] - phat[I])*DX(I, J)*weigth / D / D;
						sum2 = sum2 + (pnew[J] - phat[I])*DY(I, J)*weigth / D / D;
						sum11 = sum11 + (pnew[J] - phat[I])*DZ(I, J)*weigth / D / D;

						sum3 = sum3 + weigth*(u[J] - u[I])*NEU;
						sum4 = sum4 + weigth*(v[J] - v[I])*NEU;
						sum12 = sum12 + weigth*(w[J] - w[I])*NEU;
						if (sum3 * 0 != 0) {
							//	printf("prediction,%f,%f,%f, %f,%f,%f,%f, %f,%d,%d\n", weigth, D, u[I], u[J], NEU, MEU[I], MEU[J], NEUt[I], I, J); getchar();
						}


						//		sum5 = sum5 + weigth*(u[J] - u[I]);
						//		sum6 = sum6 + weigth*(v[J] - v[I]);
						//		sum13 = sum13 + weigth*(w[J] - w[I]);



						/*		sum7 = sum7 + (Tau_xx[J] - Tau_xx[I])*DX(I, J)*weigth / D / D;
						sum8 = sum8 + (Tau_yy[J] - Tau_yy[I])*DY(I, J)*weigth / D / D;


						sum9 = sum9 + (Tau_xy[J] - Tau_xy[I])*DX(I, J)*weigth / D / D;
						sum10 = sum10 + (Tau_xy[J] - Tau_xy[I])*DY(I, J)*weigth / D / D;*/



					}

				}

				//	if (stress_cal_method == 1)
				//	{

				ustar[I] = u[I] + gx*DT + 2 * DIM * DT*(sum3) / (lambda*n0) - (1 - relaxp)*(DIM*DT / n0 / RHO[I])*sum1 + DT*SFX[I] / RHO[I];
				vstar[I] = v[I] + gy*DT + 2 * DIM  * DT*(sum4) / (lambda*n0) - (1 - relaxp)*(DIM*DT / n0 / RHO[I])*sum2 + DT*SFY[I] / RHO[I];
				wstar[I] = w[I] + gz*DT + 2 * DIM  * DT*(sum12) / (lambda*n0) - (1 - relaxp)*(DIM*DT / n0 / RHO[I])*sum11 + DT*SFZ[I] / RHO[I];

				//	}
				//	else
				//	{
				// this part is not 3D
				//		ustar[I] = u[I] + gx*DT - (1 - relaxp)*(DIM*DT / n0 / RHO[I])*sum1 + DT*SFX[I] / RHO[I] + (2.0*DT / n0 / RHO[I])* (sum7 + sum10);
				//		vstar[I] = v[I] + gy*DT - (1 - relaxp)*(DIM*DT / n0 / RHO[I])*sum2 + DT*SFY[I] / RHO[I] + (2.0*DT / n0 / RHO[I])* (sum8 + sum9);

				//	}

				xstar[I] = x[I] + DT*ustar[I];
				ystar[I] = y[I] + DT*vstar[I];
				zstar[I] = z[I] + DT*wstar[I];




			}

		}

		//	 VISCOSITY(ustar,vstar);
		//	 if (visc_error<0.01*visc_ave) visc_itr=visc_itr_num;
		//	 }




		//------------ Particle collision and calculation of new particle number density-------

		if (int(Tstep / 2) == 1.0 / 2 * Tstep)
		{
			for (I = 1; I <= NUM; I++)
			{
				COLLISION(I, coll);
			}
		}
		else
		{
			for (I = NUM; I >= 1; I--)
			{
				COLLISION(I, coll);
			}

		}


		for (I = 1; I <= NUM; I++)
		{

			p_rheo[I] = p_rheo_new[I];
			nstar[I] = PNUMSTAR(I, KTYPE, 2, NUM);


		}
		//-------------------------------------Pressure calculation--------------------------------------

		PRESSURECALC();
		//------------------------------------Correction----------------------------------------

		for (I = 1; I <= GP + WP; I++)
		{
			BC(WBC);
		}

		//******************** Calculation of Phat ****************
		for (I = 1; I <= NUM; I++)
		{
			double min = 999999999999999;
			for (l = 2; l <= neighb[I][1]; l++)
			{
				J = neighb[I][l];

				if (pnew[J]<min)min = pnew[J];
			}
			phat[I] = min;
		}
		//**************** Calculation of pressure gradient *****************

		for (I = GP + WP + 1; I <= NUM; I++)
		{

			sum1 = 0;
			sum2 = 0;
			sum3 = 0;
			sum4 = 0;
			sum5 = 0;
			sum6 = 0;


			if (KHcorrection == 1)
			{
				for (l = 2; l <= neighb[I][1]; l++)
				{
					J = neighb[I][l];
					D = DISTSTAR(I, J);
					if (I != J && D <= re)
					{
						sum1 = sum1 + (pnew[J] + pnew[I] - 2.0*phat[I])*DXSTAR(I, J)*W(D, KTYPE, 2) / D / D;
						sum2 = sum2 + (pnew[J] + pnew[I] - 2.0*phat[I])*DYSTAR(I, J)*W(D, KTYPE, 2) / D / D;
						sum3 = sum3 + (pnew[J] + pnew[I] - 2.0*phat[I])*DZSTAR(I, J)*W(D, KTYPE, 2) / D / D;
					}
				}
			}
			else
			{
				for (l = 2; l <= neighb[I][1]; l++)
				{
					J = neighb[I][l];
					D = DISTSTAR(I, J);
					if (I != J && D <= re)
					{

						sum1 = sum1 + (pnew[J] - phat[I])*DXSTAR(I, J)*W(D, KTYPE, 2) / D / D;
						sum2 = sum2 + (pnew[J] - phat[I])*DYSTAR(I, J)*W(D, KTYPE, 2) / D / D;
						sum3 = sum3 + (pnew[J] - phat[I])*DZSTAR(I, J)*W(D, KTYPE, 2) / D / D;
					}
				}
			}
			Rho = RHO[I];
			unew[I] = ustar[I] - relaxp*(DIM*DT / n0 / Rho)*sum1;
			vnew[I] = vstar[I] - relaxp*(DIM*DT / n0 / Rho)*sum2;
			wnew[I] = wstar[I] - relaxp*(DIM*DT / n0 / Rho)*sum3;


			//----------- Damper ----------------------------

			if (fabs(unew[I])>2.0*VMAX) unew[I] = VMAX;
			if (vnew[I]>2.0*VMAX) vnew[I] = 2.0*VMAX;
			if (vnew[I]<-2.0*VMAX) vnew[I] = -2.0*VMAX;

			//------------------------------------------------

		}
		//---------------------------------- Moving particles -----------------------------------
		for (I = GP + WP + 1; I <= NUM; I++)
		{

			x[I] = x[I] + unew[I] * DT;
			y[I] = y[I] + vnew[I] * DT;
			z[I] = z[I] + wnew[I] * DT;
			BOUNDARIE();                       // Avoid penetration of particle to boundaries

		}

		//---------------------------------- Inflow ---------------------------------------------------


		if (Inflow_type>0)
		{
			INFLOW();
			SORT();
		}
		//-----------------------------------Aplying the pair-wise Collision ----------------------------------------------

		if (int(Tstep / 2) == 1.0 / 2 * Tstep)
		{
			for (I = 1; I <= NUM; I++)
			{
				COLLISION2(I, coll);
			}
		}
		else
		{
			for (I = NUM; I >= 1; I--)
			{
				COLLISION2(I, coll);
			}

		}


		//--------------------------------- print results ---------------------------------------------
		/*		if (((Tstep-1)/100.000)==int((Tstep-1)/100.000))
		{


		//	out<<" t= "<<t<<"\n";
		//	out<<"------------------------------------------------------------------\n";
		//	out<<"  I     x      y       u       v       p\n";
		//	out<<"-------------------------------------------------------------------\n";
		out<<t<<"\n";
		out<<NUM<<"\n";

		for (I=1;I<=NUM;I++)
		{

		out<<x[I]<<"  "<<y[I]<<"  "<<PTYPE[I]<<"  "<<unew[I]<<"  "<<vnew[I]<<"  "<<pnew[I]<<"\n";
		}
		}


		*/



		if (int(t * 100) == int(tec_out_intrval*tec_counter * 100))
		{

			tecplot_out << "VARIABLES  =  x,y,z,ptype,u,v,w,p,prheo,c, log_Meu, Meu_y\n";
			tecplot_out << " ZONE T=\"" << tec_out_intrval*tec_counter << "\"\n";

			for (I = 1; I <= NUM; I++)
			{


				if (PTYPE[I] >= 0)  tecplot_out << x[I] << "  " << y[I] << "	" << z[I] << "	" << PTYPE[I] << "  " << unew[I] << "  " << vnew[I] << " " << wnew[I] << " " << pnew[I] << "	" << p_rheo[I] << "  " << C[I] << "	" << log10(MEU[I]) << "	" << MEU_Y[I] << "\n";
			}
			tecplot_out << "\n";
			tec_counter++;
		}


		/*

		if (int(t*100)==int(output_time1*100) && p_count==1 )
		{
		out1<<t<<"\n";p_count++;
		for (I=1;I<=NUM;I++)
		out1<<x[I]<<"  "<<y[I]<<"  "<<PTYPE[I]<<"  "<<unew[I]<<"  "<<vnew[I]<<"  "<<pnew[I]<<"  "<<C[I]<<"\n";
		out1.close();
		}
		if (int(t*100)==int(output_time2*100) && p_count==2 )
		{
		out2<<t<<"\n";p_count++;
		for (I=1;I<=NUM;I++)
		out2<<x[I]<<"  "<<y[I]<<"  "<<PTYPE[I]<<"  "<<unew[I]<<"  "<<vnew[I]<<"  "<<pnew[I]<<"  "<<C[I]<<"\n";
		out2.close();
		}
		if (int(t*100)==int(output_time3*100) && p_count==3 )
		{
		out3<<t<<"\n";p_count++;
		for (I=1;I<=NUM;I++)
		out3<<x[I]<<"  "<<y[I]<<"  "<<PTYPE[I]<<"  "<<unew[I]<<"  "<<vnew[I]<<"  "<<pnew[I]<<"  "<<C[I]<<"\n";
		out3.close();
		}
		if (int(t*100)==int(output_time4*100) && p_count==4 )
		{
		out4<<t<<"\n";p_count++;
		for (I=1;I<=NUM;I++)
		out4<<x[I]<<"  "<<y[I]<<"  "<<PTYPE[I]<<"  "<<unew[I]<<"  "<<vnew[I]<<"  "<<pnew[I]<<"  "<<C[I]<<"\n";
		out4.close();
		}
		if (int(t*100)==int(output_time5*100) && p_count==5 )
		{
		out5<<t<<"\n";p_count++;
		for (I=1;I<=NUM;I++)
		out5<<x[I]<<"  "<<y[I]<<"  "<<PTYPE[I]<<"  "<<unew[I]<<"  "<<vnew[I]<<"  "<<pnew[I]<<"  "<<C[I]<<"\n";
		out5.close();
		}
		if (int(t*100)==int(output_time6*100) && p_count==6 )
		{
		out6<<t<<"\n";p_count++;
		for (I=1;I<=NUM;I++)
		out6<<x[I]<<"  "<<y[I]<<"  "<<PTYPE[I]<<"  "<<unew[I]<<"  "<<vnew[I]<<"  "<<pnew[I]<<"  "<<C[I]<<"\n";
		out6.close();
		}
		if (int(t*100)==int(output_time7*100) && p_count==7 )
		{
		out7<<t<<"\n";p_count++;
		for (I=1;I<=NUM;I++)
		out7<<x[I]<<"  "<<y[I]<<"  "<<PTYPE[I]<<"  "<<unew[I]<<"  "<<vnew[I]<<"  "<<pnew[I]<<"  "<<C[I]<<"\n";
		out7.close();
		}
		if (int(t*100)==int(output_time8*100) && p_count==8 )
		{
		out8<<t<<"\n";p_count++;
		for (I=1;I<=NUM;I++)
		out8<<x[I]<<"  "<<y[I]<<"  "<<PTYPE[I]<<"  "<<unew[I]<<"  "<<vnew[I]<<"  "<<pnew[I]<<"  "<<C[I]<<"\n";
		out8.close();
		}
		if (int(t*100)==int(output_time9*100) && p_count==9 )
		{
		out9<<t<<"\n";p_count++;
		for (I=1;I<=NUM;I++)
		out9<<x[I]<<"  "<<y[I]<<"  "<<PTYPE[I]<<"  "<<unew[I]<<"  "<<vnew[I]<<"  "<<pnew[I]<<"  "<<C[I]<<"\n";
		out9.close();
		}

		if (int(t*100)==int(output_time10*100) && p_count==10 )
		{
		out10<<t<<"\n";p_count++;
		for (I=1;I<=NUM;I++)
		out10<<x[I]<<"  "<<y[I]<<"  "<<PTYPE[I]<<"  "<<unew[I]<<"  "<<vnew[I]<<"  "<<pnew[I]<<"  "<<C[I]<<"\n";
		out10.close();
		}
		if (int(t*100)==int(output_time11*100) && p_count==11 )
		{
		out11<<t<<"\n";p_count++;
		for (I=1;I<=NUM;I++)
		out11<<x[I]<<"  "<<y[I]<<"  "<<PTYPE[I]<<"  "<<unew[I]<<"  "<<vnew[I]<<"  "<<pnew[I]<<"  "<<C[I]<<"\n";
		out11.close();
		}
		if (int(t*100)==int(output_time12*100) && p_count==12 )
		{
		out12<<t<<"\n";p_count++;
		for (I=1;I<=NUM;I++)
		out12<<x[I]<<"  "<<y[I]<<"  "<<PTYPE[I]<<"  "<<unew[I]<<"  "<<vnew[I]<<"  "<<pnew[I]<<"  "<<C[I]<<"\n";
		out12.close();
		}
		if (int(t*100)==int(output_time13*100) && p_count==13 )
		{
		out13<<t<<"\n";p_count++;
		for (I=1;I<=NUM;I++)
		out13<<x[I]<<"  "<<y[I]<<"  "<<PTYPE[I]<<"  "<<unew[I]<<"  "<<vnew[I]<<"  "<<pnew[I]<<"  "<<C[I]<<"\n";
		out13.close();
		}
		if (int(t*100)==int(output_time14*100) && p_count==14 )
		{
		out14<<t<<"\n";p_count++;
		for (I=1;I<=NUM;I++)
		out14<<x[I]<<"  "<<y[I]<<"  "<<PTYPE[I]<<"  "<<unew[I]<<"  "<<vnew[I]<<"  "<<pnew[I]<<"  "<<C[I]<<"\n";
		out14.close();
		}
		if (int(t*100)==int(output_time15*100) && p_count==15 )
		{
		out15<<t<<"\n";p_count++;
		for (I=1;I<=NUM;I++)
		out15<<x[I]<<"  "<<y[I]<<"  "<<PTYPE[I]<<"  "<<unew[I]<<"  "<<vnew[I]<<"  "<<pnew[I]<<"  "<<C[I]<<"\n";
		out15.close();
		}
		if (int(t*100)==int(output_time16*100) && p_count==16 )
		{
		out16<<t<<"\n";p_count++;
		for (I=1;I<=NUM;I++)
		out16<<x[I]<<"  "<<y[I]<<"  "<<PTYPE[I]<<"  "<<unew[I]<<"  "<<vnew[I]<<"  "<<pnew[I]<<"  "<<C[I]<<"\n";
		out16.close();
		}
		if (int(t*100)==int(output_time17*100) && p_count==17 )
		{
		out17<<t<<"\n";p_count++;
		for (I=1;I<=NUM;I++)
		out17<<x[I]<<"  "<<y[I]<<"  "<<PTYPE[I]<<"  "<<unew[I]<<"  "<<vnew[I]<<"  "<<pnew[I]<<"  "<<C[I]<<"\n";
		out17.close();
		}
		if (int(t*100)==int(output_time18*100) && p_count==18 )
		{
		out18<<t<<"\n";p_count++;
		for (I=1;I<=NUM;I++)
		out18<<x[I]<<"  "<<y[I]<<"  "<<PTYPE[I]<<"  "<<unew[I]<<"  "<<vnew[I]<<"  "<<pnew[I]<<"  "<<C[I]<<"\n";
		out18.close();
		}
		if (int(t*100)==int(output_time19*100) && p_count==19 )
		{
		out19<<t<<"\n";p_count++;
		for (I=1;I<=NUM;I++)
		out19<<x[I]<<"  "<<y[I]<<"  "<<PTYPE[I]<<"  "<<unew[I]<<"  "<<vnew[I]<<"  "<<pnew[I]<<"  "<<C[I]<<"\n";
		out19.close();
		}

		if (int(t*100)==int(output_time20*100) && p_count==20 )
		{
		out20<<t<<"\n";p_count++;
		for (I=1;I<=NUM;I++)
		out20<<x[I]<<"  "<<y[I]<<"  "<<PTYPE[I]<<"  "<<unew[I]<<"  "<<vnew[I]<<"  "<<pnew[I]<<"  "<<C[I]<<"\n";
		out20.close();
		}
		if (int(t*100)==int(output_time21*100) && p_count==21 )
		{
		out21<<t<<"\n";p_count++;
		for (I=1;I<=NUM;I++)
		out21<<x[I]<<"  "<<y[I]<<"  "<<PTYPE[I]<<"  "<<unew[I]<<"  "<<vnew[I]<<"  "<<pnew[I]<<"  "<<C[I]<<"\n";
		out21.close();
		}
		if (int(t*100)==int(output_time22*100) && p_count==22 )
		{
		out22<<t<<"\n";p_count++;
		for (I=1;I<=NUM;I++)
		out22<<x[I]<<"  "<<y[I]<<"  "<<PTYPE[I]<<"  "<<unew[I]<<"  "<<vnew[I]<<"  "<<pnew[I]<<"  "<<C[I]<<"\n";
		out22.close();
		}
		if (int(t*100)==int(output_time23*100) && p_count==23 )
		{
		out23<<t<<"\n";p_count++;
		for (I=1;I<=NUM;I++)
		out23<<x[I]<<"  "<<y[I]<<"  "<<PTYPE[I]<<"  "<<unew[I]<<"  "<<vnew[I]<<"  "<<pnew[I]<<"  "<<C[I]<<"\n";
		out23.close();
		}
		if (int(t*100)==int(output_time24*100) && p_count==24 )
		{
		out24<<t<<"\n";p_count++;
		for (I=1;I<=NUM;I++)
		out24<<x[I]<<"  "<<y[I]<<"  "<<PTYPE[I]<<"  "<<unew[I]<<"  "<<vnew[I]<<"  "<<pnew[I]<<"  "<<C[I]<<"\n";
		out24.close();
		}
		if (int(t*100)==int(output_time25*100) && p_count==25 )
		{
		out25<<t<<"\n";p_count++;
		for (I=1;I<=NUM;I++)
		out25<<x[I]<<"  "<<y[I]<<"  "<<PTYPE[I]<<"  "<<unew[I]<<"  "<<vnew[I]<<"  "<<pnew[I]<<"  "<<C[I]<<"\n";
		out25.close();
		}
		if (int(t*100)==int(output_time26*100) && p_count==26 )
		{
		out26<<t<<"\n";p_count++;
		for (I=1;I<=NUM;I++)
		out26<<x[I]<<"  "<<y[I]<<"  "<<PTYPE[I]<<"  "<<unew[I]<<"  "<<vnew[I]<<"  "<<pnew[I]<<"  "<<C[I]<<"\n";
		out26.close();
		}
		if (int(t*100)==int(output_time27*100) && p_count==27 )
		{
		out27<<t<<"\n";p_count++;
		for (I=1;I<=NUM;I++)
		out27<<x[I]<<"  "<<y[I]<<"  "<<PTYPE[I]<<"  "<<unew[I]<<"  "<<vnew[I]<<"  "<<pnew[I]<<"  "<<C[I]<<"\n";
		out27.close();
		}
		if (int(t*100)==int(output_time28*100) && p_count==28 )
		{
		out28<<t<<"\n";p_count++;
		for (I=1;I<=NUM;I++)
		out28<<x[I]<<"  "<<y[I]<<"  "<<PTYPE[I]<<"  "<<unew[I]<<"  "<<vnew[I]<<"  "<<pnew[I]<<"  "<<C[I]<<"\n";
		out28.close();
		}
		if (int(t*100)==int(output_time29*100) && p_count==29 )
		{
		out29<<t<<"\n";p_count++;
		for (I=1;I<=NUM;I++)
		out29<<x[I]<<"  "<<y[I]<<"  "<<PTYPE[I]<<"  "<<unew[I]<<"  "<<vnew[I]<<"  "<<pnew[I]<<"  "<<C[I]<<"\n";
		out29.close();
		}

		if (int(t*100)==int(output_time30*100) && p_count==30 )
		{
		out30<<t<<"\n";p_count++;
		for (I=1;I<=NUM;I++)
		out30<<x[I]<<"  "<<y[I]<<"  "<<PTYPE[I]<<"  "<<unew[I]<<"  "<<vnew[I]<<"  "<<pnew[I]<<"  "<<C[I]<<"\n";
		out30.close();
		}
		if (int(t*100)==int(output_time31*100) && p_count==31 )
		{
		out31<<t<<"\n";p_count++;
		for (I=1;I<=NUM;I++)
		out31<<x[I]<<"  "<<y[I]<<"  "<<PTYPE[I]<<"  "<<unew[I]<<"  "<<vnew[I]<<"  "<<pnew[I]<<"  "<<C[I]<<"\n";
		out31.close();
		}
		if (int(t*100)==int(output_time32*100) && p_count==32 )
		{
		out32<<t<<"\n";p_count++;
		for (I=1;I<=NUM;I++)
		out32<<x[I]<<"  "<<y[I]<<"  "<<PTYPE[I]<<"  "<<unew[I]<<"  "<<vnew[I]<<"  "<<pnew[I]<<"  "<<C[I]<<"\n";
		out32.close();
		}
		if (int(t*100)==int(output_time33*100) && p_count==33 )
		{
		out33<<t<<"\n";p_count++;
		for (I=1;I<=NUM;I++)
		out33<<x[I]<<"  "<<y[I]<<"  "<<PTYPE[I]<<"  "<<unew[I]<<"  "<<vnew[I]<<"  "<<pnew[I]<<"  "<<C[I]<<"\n";
		out33.close();
		}

		if (int(t*100)==int(output_time34*100) && p_count==34 )
		{
		out34<<t<<"\n";p_count++;
		for (I=1;I<=NUM;I++)
		out34<<x[I]<<"  "<<y[I]<<"  "<<PTYPE[I]<<"  "<<unew[I]<<"  "<<vnew[I]<<"  "<<pnew[I]<<"  "<<C[I]<<"\n";
		out34.close();
		}
		if (int(t*100)==int(output_time35*100) && p_count==35 )
		{
		out35<<t<<"\n";p_count++;
		for (I=1;I<=NUM;I++)
		out35<<x[I]<<"  "<<y[I]<<"  "<<PTYPE[I]<<"  "<<unew[I]<<"  "<<vnew[I]<<"  "<<pnew[I]<<"  "<<C[I]<<"\n";
		out35.close();
		}
		if (int(t*100)==int(output_time36*100) && p_count==36 )
		{
		out36<<t<<"\n";p_count++;
		for (I=1;I<=NUM;I++)
		out36<<x[I]<<"  "<<y[I]<<"  "<<PTYPE[I]<<"  "<<unew[I]<<"  "<<vnew[I]<<"  "<<pnew[I]<<"  "<<C[I]<<"\n";
		out36.close();
		}
		if (int(t*100)==int(output_time37*100) && p_count==37 )
		{
		out37<<t<<"\n";p_count++;
		for (I=1;I<=NUM;I++)
		out37<<x[I]<<"  "<<y[I]<<"  "<<PTYPE[I]<<"  "<<unew[I]<<"  "<<vnew[I]<<"  "<<pnew[I]<<"  "<<C[I]<<"\n";
		out37.close();
		}
		if (int(t*100)==int(output_time38*100) && p_count==38 )
		{
		out38<<t<<"\n";p_count++;
		for (I=1;I<=NUM;I++)
		out38<<x[I]<<"  "<<y[I]<<"  "<<PTYPE[I]<<"  "<<unew[I]<<"  "<<vnew[I]<<"  "<<pnew[I]<<"  "<<C[I]<<"\n";
		out38.close();
		}
		if (int(t*100)==int(output_time39*100) && p_count==39 )
		{
		out39<<t<<"\n";p_count++;
		for (I=1;I<=NUM;I++)
		out39<<x[I]<<"  "<<y[I]<<"  "<<PTYPE[I]<<"  "<<unew[I]<<"  "<<vnew[I]<<"  "<<pnew[I]<<"  "<<C[I]<<"\n";
		out39.close();
		}

		if (int(t*100)==int(output_time40*100) && p_count==40 )
		{
		out40<<t<<"\n";p_count++;
		for (I=1;I<=NUM;I++)
		out40<<x[I]<<"  "<<y[I]<<"  "<<PTYPE[I]<<"  "<<unew[I]<<"  "<<vnew[I]<<"  "<<pnew[I]<<"  "<<C[I]<<"\n";
		out40.close();
		}*/


		//--------------------- Prepare data for new time step----------------------------------------------------------

		for (I = 1; I <= NUM; I++)
		{
			p[I] = pnew[I];
			u[I] = unew[I];
			v[I] = vnew[I];
			w[I] = wnew[I];


			if (p[I] * 0 != 0 || u[I] * 0 != 0 || v[I] * 0 != 0)
			{
				//cout<<"Error Check"<<p[I]<<" "<<u[I]<< " "<<v[I]<<endl;
				//int a;
				//cin>>a;
				printf("ERROR#1: ERROR in particle %d , x=%f, y=%f, p=%f\n", I, x[I], y[I], p[I]);
				getchar();
			}

			if (x[I] >= Xmax || y[I] >= Ymax || z[I] >= Zmax || x[I] <= Xmin || y[I] <= Ymin || z[I] <= Zmin)
			{
				printf("ERROR#2: ERROR in particle %d , x=%f, y=%f\n", I, x[I], y[I], z[I]);
			}
		}


		t = t + DT;

		if (Tstep == 1)
		{
			time(&end);
			dif = difftime(end, start);

			printf("Estimated running time:%.2fsec (per time step)\n", dif);
			printf("                                      OR %.2f hr (per 1sec simulation)\n", dif / DT / 3600);
			printf("                  ", t);


		}
		printf("|");


	}
	// --------------------------------End of Time Loop --------------------------------------------------


	return;
}

