
#include <hip/hip_runtime.h>
#include<algorithm>
#include <stdio.h>
#include <math.h>
#include <fstream>
#include <iostream>
#include <time.h>
#include <vector>

using namespace std;

double x[10]={1,2,3,4,5,6,7,8,9,0};
double y[10]={1,2,3,4,5,6,7,8,9,0};
double z[10]={1,2,3,4,5,6,7,8,9,0};
double Xmax=9, Xmin=0;
double Ymax=9, Ymin=0;
double Zmax=9, Zmin=0;
double re=1, DELTA=0;
int NUM=100;


//declarations of global variables such as x, y, z, re, DELTA, xmax, etc/ 

//CUDA kernel for calcHash
__global__ void calcHash(double *d_x, double *d_y, double *d_z, uint2 *d_particleHash,\
	int *d_NUM, double *d_Xmax, double *d_Xmin, double *d_re, double *d_DELTA, double *d_Ymin, double *d_Ymax, double *d_Zmax, double *d_Zmin){

  int ncx, ncy, ncz;
  ncx = int((*d_Xmax - *d_Xmin) / (*d_re + *d_DELTA)) + 1;     // Number of cells in x direction
  ncy = int((*d_Ymax - *d_Ymin) / (*d_re + *d_DELTA)) + 1;     // Number of cells in y direction
  ncz = int((*d_Zmax - *d_Zmin) / (*d_re + *d_DELTA)) + 1;     // Number of cells in z direction

  int k = threadIdx.x + blockIdx.x * blockDim.x;
  int m = *d_NUM;
  int *icell, *jcell, *kcell;

  icell[k] = int((d_x[k] - *d_Xmin) / (*d_re + *d_DELTA)) + 1;
  jcell[k] = int((d_y[k] - *d_Ymin) / (*d_re + *d_DELTA)) + 1;
  kcell[k] = int((d_z[k] - *d_Zmin) / (*d_re + *d_DELTA)) + 1;

  int cellNum = icell[k] + (jcell[k] - 1)*ncx + (kcell[k] - 1)*ncx*ncy;

  d_particleHash[k].x = cellNum;
  d_particleHash[k].y = k;

}


/*&CUDA kernel for findCellStart

__global__ void findCellStart(int *particleHash, int *cellStart, int *cellEnd){

  int k = 1 + threadIdx.x + blockIdx.x * blockDim.x; //take care of index shift in # of threads
  if (particleHash[k] != particleHash[k+1]){
    cellEnd[particleHash[k]] = k;
    cellStart[particleHash[k+1]] = k+1;
  }

}
//CUDA kernel for neighb[][] creation

__global__ void createNeighbourArrays(int **neighb, int *cellStart, int *cellEnd, int *particleHash){

  //check which type of cell it is and loop surrounding cells
  loop{
    add the particles in those cells as neighbours in the 2-d array
  }

}
*/


void neighbour_cuda(){

  int *d_cellEnd, *d_cellStart, *d_NUM;
  uint2 *particleHash, *d_particleHash;
  double *d_x, *d_y, *d_z, *d_Xmax, *d_Xmin, *d_Ymax, *d_Ymin, *d_Zmax, *d_Zmin, *d_re, *d_DELTA;

  int arrsizeint = NUM * sizeof(int);
  int sizeint = sizeof(int);
  int arrsizedouble = NUM * sizeof(double);
  int sizedouble = sizeof(double);
  int arruint2size = NUM * sizeof(uint2);

  particleHash = (uint2 *)malloc(arruint2size);


  hipMalloc((void **)&d_particleHash, arruint2size);
  hipMalloc((void **)&d_cellStart, arrsizeint);
  hipMalloc((void **)&d_cellEnd, arrsizeint);
  hipMalloc((void **)&d_x, arrsizedouble);
  hipMalloc((void **)&d_y, arrsizedouble);
  hipMalloc((void **)&d_z, arrsizedouble);
  hipMalloc((void **)&d_Xmin, sizedouble);
  hipMalloc((void **)&d_Xmax, sizedouble);
  hipMalloc((void **)&d_Ymin, sizedouble);
  hipMalloc((void **)&d_Ymax, sizedouble);
  hipMalloc((void **)&d_Zmin, sizedouble);
  hipMalloc((void **)&d_Zmax, sizedouble);
  hipMalloc((void **)&d_re, sizedouble);
  hipMalloc((void **)&d_DELTA, sizedouble);
  hipMalloc((void **)&d_NUM, sizeint);

  //cudaMemcpy(d_particleHash, particleHash, arrsizeint, cudaMemcpyHostToDevice);
  //cudaMemcpy(d_cellStart, cellStart, arrsizeint, cudaMemcpyHostToDevice);
  //cudaMemcpy(d_cellEnd, cellEnd, arrsizeint, cudaMemcpyHostToDevice);
  hipMemcpy(d_x, &x, arrsizedouble, hipMemcpyHostToDevice);
  hipMemcpy(d_y, &y, arrsizedouble, hipMemcpyHostToDevice);
  hipMemcpy(d_z, &z, arrsizedouble, hipMemcpyHostToDevice);
  hipMemcpy(d_Xmin, &Xmin, sizedouble, hipMemcpyHostToDevice);
  hipMemcpy(d_Xmax, &Xmax, sizedouble, hipMemcpyHostToDevice);
  hipMemcpy(d_Ymin, &Ymin, sizedouble, hipMemcpyHostToDevice);
  hipMemcpy(d_Ymax, &Ymax, sizedouble, hipMemcpyHostToDevice);
  hipMemcpy(d_Zmin, &Zmin, sizedouble, hipMemcpyHostToDevice);
  hipMemcpy(d_Zmax, &Zmax, sizedouble, hipMemcpyHostToDevice);
  hipMemcpy(d_re, &re, sizedouble, hipMemcpyHostToDevice);
  hipMemcpy(d_DELTA, &DELTA, sizedouble, hipMemcpyHostToDevice);
  hipMemcpy(d_NUM, &NUM, sizeint, hipMemcpyHostToDevice);




  //copy x, y, z onto the device
  //run calc hash on the global array
  calcHash<<<5,5>>>(d_x, d_y, d_z, d_particleHash, d_NUM, d_Xmax, d_Xmin, d_re, d_DELTA, d_Ymin, d_Ymax, d_Zmax, d_Zmin);
  hipMemcpy(particleHash, d_particleHash, arruint2size, hipMemcpyDeviceToHost);
  for(int k=0; k<10; k++){
  cout<<particleHash[k].x<<"  "<<particleHash[k].y<<endl;
  }

  //radixsort - the particleHash array 
  //run findcellstart on the global array - Ista and Iend - check collision
  //findcellstart<<<NUM/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(particleHash);
  //create the neighbour arrays for each particle
  //createNeighbourArrays<<<NUM/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(cellstart, cellend);

  hipFree(d_particleHash);
  hipFree(d_cellStart);
  hipFree(d_cellEnd);
  hipFree(d_x);
  hipFree(d_y);
  hipFree(d_z);
  hipFree(d_Xmin);
  hipFree(d_Xmax);
  hipFree(d_Ymin);
  hipFree(d_Ymax);
  hipFree(d_Zmin);
  hipFree(d_Zmax);
  hipFree(d_re);
  hipFree(d_NUM);

}
/*

void NEIGHBOR()
{

	// ------------------PARAMETERS DEFENTION -------------------------------------
	int ncx = int((Xmax - Xmin) / (re + DELTA)) + 1;     // Number of cells in x direction
	int ncy = int((Ymax - Ymin) / (re + DELTA)) + 1;     // Number of cells in y direction
	int ncz = int((Zmax - Zmin) / (re + DELTA)) + 1;     // Number of cells in z direction

	int tnc = ncx*ncy*ncz;							   // Total number of cells   
	int m, k, kmax, Cnum;


	int *Ista, *Iend, *nc, *icell, *jcell, *kcell;
	int *ip;                             // I is sorted number of ip[I] th paricle
	Ista = new int[tnc + 1]; //this points to the index of the first element in a cell in the array ip
	Iend = new int[tnc + 1]; //index of the last element in a cell in the array ip
	nc = new int[tnc + 1];
	icell = new int[TP + 1];
	jcell = new int[TP + 1];
	kcell = new int[TP + 1];
	ip = new int[TP + 1]; //this is the main array that we are looking for, it is sorted 
  // according to cell numbers and it contains particle indices 



	//----------------- ALLOCATING PRTICLES IN CELLS --------------------------


	for (k = 1; k <= tnc; k++) //cell loop 
	{
		Ista[k] = 1;
		Iend[k] = 0;
		nc[k] = 0;
	}

	for (k = 1; k <= NUM; k++) //particle loop
	{
		icell[k] = int((x[k] - Xmin) / (re + DELTA)) + 1;
		jcell[k] = int((y[k] - Ymin) / (re + DELTA)) + 1;
		kcell[k] = int((z[k] - Zmin) / (re + DELTA)) + 1;

		Cnum = icell[k] + (jcell[k] - 1)*ncx + (kcell[k] - 1)*ncx*ncy;     // Cell number in which particle k located

		nc[Cnum]++;						            // Number of particle in cell Cnum
		Iend[Cnum]++;						        // Number of particle in cell Cnum 

		for (m = Iend[tnc]; m >= Iend[Cnum]; m--)
		{
			if (m>0) ip[m + 1] = ip[m];
		} //this block is there to create space at the end as and when new particles are added

		for (m = Cnum + 1; m <= tnc; m++)
		{
			Ista[m]++;
			Iend[m]++;
		}

		ip[Iend[Cnum]] = k;
	}


	//--------------- FINDIND NEIGHBORS ----------------------------------
	int JJ;
	for (I = 1; I <= NUM; I++)
	{
		k = 2;
		int row, colu, elev, m1, m2, m3, m4, m5, m6;
		if (icell[I] == 1)m1 = 0; else m1 = -1;
		if (icell[I] == ncx)m2 = 0; else m2 = +1;
		if (jcell[I] == 1)m3 = 0; else m3 = -1;
		if (jcell[I] == ncy)m4 = 0; else m4 = +1;
		if (kcell[I] == 1)m5 = 0; else m5 = -1;
		if (kcell[I] == ncz)m6 = 0; else m6 = +1;

		for (row = m1; row <= m2; row++) //could be -1 to 1 , the triple loop is basically there to find all the 9 cells around that particle, including the one in which it itself is
		{
			for (colu = m3; colu <= m4; colu++) 
			{
				for (elev = m5; elev <= m6; elev++)
				{

					Cnum = icell[I] + row + (jcell[I] - 1 + colu)*ncx + (kcell[I] - 1 + elev)*ncx*ncy;

					for (JJ = Ista[Cnum]; JJ <= Iend[Cnum]; JJ++)
					{
						J = ip[JJ]; //J is tha ACTUAL particle index 
						neighb[I][k] = J;
						k++;
					}
				}
			}
		}
		kmax = k - 2;
		neighb[I][1] = kmax; //this is the total number of neighbours, which is stored at the beginning 
		//if( neighb[I][1]>1098 ||neighb[I][1]*0!=0) printf("ERROR, the neighbors of particles %d is %d", I, neighb[I][1]);
	}
	//--------------------Clearing dynamic arrays ----------------------------

	delete[]Ista;
	delete[]Iend;
	delete[]nc;
	delete[]icell;
	delete[]jcell;
	delete[]kcell;
	delete[]ip;
	Ista = NULL; Iend = NULL; nc = NULL; icell = NULL; jcell = NULL; kcell = NULL, ip = NULL;
}

*/


int main(){

	neighbour_cuda();
	return 0;
}