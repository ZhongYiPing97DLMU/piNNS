#include "hip/hip_runtime.h"
#include <algorithm>
#include <stdio.h>
#include <math.h>
#include <fstream>
#include <iostream>
#include <time.h>
#include <vector>
#include <thrust/sort.h>

using namespace std;

double x[10]={1,1.001,3,4,5,6,7,8,9,0};
double y[10]={1,1.001,3,4,5,6,7,8,9,0};
double z[10]={1,1.001,3,4,5,6,7,8,9,0};
double Xmax=9, Xmin=0;
double Ymax=9, Ymin=0;
double Zmax=9, Zmin=0;
double re=4, DELTA=0;
int NUM=10;
int MAX_NEIGHB=100;
int **neighb;
int *particleHash, *particleid, *cellStart, *cellEnd;

// ----------------- CUDA KERNELS -------------------------

__global__ void calcHash(double *d_x, double *d_y, double *d_z, int *d_particleHash,\
	int *d_NUM, double *d_Xmax, double *d_Xmin, double *d_re, double *d_DELTA, double *d_Ymin, \
  double *d_Ymax, double *d_Zmax, double *d_Zmin, int *d_particleid, int *d_tnc, int *ncx, int *ncy,\
  int *ncz){

  *ncx = int((*d_Xmax - *d_Xmin) / (*d_re + *d_DELTA)) + 1;     // Number of cells in x direction
  *ncy = int((*d_Ymax - *d_Ymin) / (*d_re + *d_DELTA)) + 1;     // Number of cells in y direction
  *ncz = int((*d_Zmax - *d_Zmin) / (*d_re + *d_DELTA)) + 1;     // Number of cells in z direction
  *d_tnc = *ncx * *ncy * *ncz;

  int k = threadIdx.x + blockIdx.x * blockDim.x;
  int *icell, *jcell, *kcell;

  int cellarrsize =   sizeof(int);
  icell = (int *)malloc(cellarrsize);
  jcell = (int *)malloc(cellarrsize);
  kcell = (int *)malloc(cellarrsize);
  
  *icell = int((d_x[k] - *d_Xmin) / (*d_re + *d_DELTA)) + 1;
  *jcell = int((d_y[k] - *d_Ymin) / (*d_re + *d_DELTA)) + 1;
  *kcell = int((d_z[k] - *d_Zmin) / (*d_re + *d_DELTA)) + 1;

  int cellNum = *icell + (*jcell - 1)* *ncx + (*kcell - 1)* *ncx * *ncy;

  d_particleHash[k] = cellNum;
  d_particleid[k] = k;

}

__global__ void findCellStart(int *particleHash, int *cellStart, int *cellEnd, int *NUM){

  int k = threadIdx.x + blockIdx.x * blockDim.x; // here index value is equal to the cell number which starts with 1 
  if (particleHash[k] != particleHash[k+1] and k!= *NUM - 1){
    cellEnd[particleHash[k] - 1] = k;
    cellStart[particleHash[k+1] - 1] = k+1;
  }
  if(k == *NUM - 1){
    cellEnd[particleHash[k] - 1] = k;
  }
              
}

__global__ void createNeighbourArraysCUDA(int *d_neighb, int *cellStart, int *cellEnd, int *particleHash, int *particleid, int *ncx, int *ncy, int *ncz, int *d_max_neighb, int *test){

  int index = threadIdx.x + blockIdx.x * blockDim.x; 
  int pid, icell, jcell, kcell, cellNum;

  cellNum = particleHash[index]; 
  pid = particleid[index];
  
  int neighb_index = pid * (*d_max_neighb + 1);

  kcell = (cellNum - 1)/((*ncx) * (*ncy)) + 1;
  jcell = ((cellNum - 1) - ((kcell - 1)* (*ncx) * (*ncy)))/ *ncx + 1;
  icell = cellNum - 1 - *ncx * (jcell - 1) - (*ncx * *ncy)*(kcell - 1) + 1;

  int Cnum, J;
  int curr_neighb_num = 0;
  
  int row, colu, elev, m1, m2, m3, m4, m5, m6;
  if (icell == 1)m1 = 0; else m1 = -1;
  if (icell == *ncx)m2 = 0; else m2 = +1;
  if (jcell == 1)m3 = 0; else m3 = -1;
  if (jcell == *ncy)m4 = 0; else m4 = +1;
  if (kcell == 1)m5 = 0; else m5 = -1;
  if (kcell == *ncz)m6 = 0; else m6 = +1;

  for (row = m1; row <= m2; row++)
  {
    for (colu = m3; colu <= m4; colu++) 
    {
      for (elev = m5; elev <= m6; elev++)
      {

        Cnum = icell + row + (jcell - 1 + colu)* *ncx + (kcell - 1 + elev)* *ncx* *ncy;

        if (cellEnd[Cnum - 1] != -1){

        for (int JJ = cellStart[Cnum -1]; JJ <= cellEnd[Cnum - 1]; JJ++)
        {
          J = particleid[JJ];
          curr_neighb_num++;
          d_neighb[neighb_index + curr_neighb_num] = J;
          
        }
      }
      }
    }
  }
  
  
  d_neighb[neighb_index] = curr_neighb_num;
  test[index] = d_neighb[neighb_index];
 
}

__global__ void InitializeCellDetails(int *cellStart, int *cellEnd){
  int index = threadIdx.x + blockIdx.x * blockDim.x; 
  cellStart[index] = 0; cellEnd[index] = -1;
}

__global__ void Template(int *particleHash, int *particleid, int *cellStart, int *cellEnd, int *ncx, int *ncy, int *ncz, int *size_neighbours, int *test){
  int index = threadIdx.x + blockDim.x * blockIdx.x;
  int pid, icell, jcell, kcell, cellNum;
  int *neighbours;
  neighbours = (int *)malloc(*size_neighbours);
  cellNum = particleHash[index]; 
  pid = particleid[index];

  kcell = (cellNum - 1)/((*ncx) * (*ncy)) + 1;
  jcell = ((cellNum - 1) - ((kcell - 1)* (*ncx) * (*ncy)))/ *ncx + 1;
  icell = cellNum - 1 - *ncx * (jcell - 1) - (*ncx * *ncy)*(kcell - 1) + 1;

  int Cnum, J;
  int curr_neighb_num = 0;
  
  int row, colu, elev, m1, m2, m3, m4, m5, m6;
  if (icell == 1)m1 = 0; else m1 = -1;
  if (icell == *ncx)m2 = 0; else m2 = +1;
  if (jcell == 1)m3 = 0; else m3 = -1;
  if (jcell == *ncy)m4 = 0; else m4 = +1;
  if (kcell == 1)m5 = 0; else m5 = -1;
  if (kcell == *ncz)m6 = 0; else m6 = +1;

  for (row = m1; row <= m2; row++)
  {
    for (colu = m3; colu <= m4; colu++) 
    {
      for (elev = m5; elev <= m6; elev++)
      {

        Cnum = icell + row + (jcell - 1 + colu)* *ncx + (kcell - 1 + elev)* *ncx* *ncy;

        if (cellEnd[Cnum - 1] != -1){

        for (int JJ = cellStart[Cnum -1]; JJ <= cellEnd[Cnum - 1]; JJ++)
        {
          J = particleid[JJ];
          curr_neighb_num++;
          neighbours[curr_neighb_num] = J;
          
        }
      }
      }
    }
  }
  
  
  neighbours[0] = curr_neighb_num;
  test[pid] = curr_neighb_num;

  //any further operations can be done using this neighbour array

}


// ------------------------- Host sub-sub-routine for neighbour computation ------------------------ 

void NEIGHBOUR_cuda(){

  // ------------------ variable declarations and initializations ------------------------------

  int *d_cellEnd, *d_cellStart, *d_NUM, *d_tnc, *tnc, *d_ncx, *d_ncy, *d_ncz, *d_max_neighb;
  int *d_particleHash, *d_particleid, *d_neighb, *h_neighb, *d_test, *test, *d_sizeof_neighbours;
  double *d_x, *d_y, *d_z, *d_Xmax, *d_Xmin, *d_Ymax, *d_Ymin, *d_Zmax, *d_Zmin, *d_re, *d_DELTA;

  int arrsizeint = NUM * sizeof(int);
  int sizeint = sizeof(int);
  int arrsizedouble = NUM * sizeof(double);
  int sizedouble = sizeof(double);
  int sizeneighb = NUM * (MAX_NEIGHB + 1) * sizeof(int);
  int sizeof_neighbours = (MAX_NEIGHB + 1) * sizeof(int);

  
  particleHash = (int *)malloc(arrsizeint);
  particleid = (int *)malloc(arrsizeint);
  tnc = (int *)malloc(sizeint);
  h_neighb = (int *)malloc(sizeneighb);

  hipMalloc((void **)&d_particleHash, arrsizeint);
  hipMalloc((void **)&d_particleid, arrsizeint); 
  
  hipMalloc((void **)&d_x, arrsizedouble);
  hipMalloc((void **)&d_y, arrsizedouble);
  hipMalloc((void **)&d_z, arrsizedouble);
  hipMalloc((void **)&d_Xmin, sizedouble);
  hipMalloc((void **)&d_Xmax, sizedouble);
  hipMalloc((void **)&d_Ymin, sizedouble);
  hipMalloc((void **)&d_Ymax, sizedouble);
  hipMalloc((void **)&d_Zmin, sizedouble);
  hipMalloc((void **)&d_Zmax, sizedouble);
  hipMalloc((void **)&d_re, sizedouble);
  hipMalloc((void **)&d_DELTA, sizedouble);
  hipMalloc((void **)&d_NUM, sizeint);
  hipMalloc((void **)&d_tnc, sizeint);
  hipMalloc((void **)&d_ncx, sizeint);
  hipMalloc((void **)&d_ncy, sizeint);
  hipMalloc((void **)&d_ncz, sizeint);
  hipMalloc((void **)&d_neighb, sizeneighb);
  hipMalloc((void **)&d_max_neighb, sizeint);
  hipMalloc((void **)&d_test, arrsizeint);
  hipMalloc((void **)&d_sizeof_neighbours, sizeof_neighbours);

  hipMemcpy(d_x, &x, arrsizedouble, hipMemcpyHostToDevice);
  hipMemcpy(d_y, &y, arrsizedouble, hipMemcpyHostToDevice);
  hipMemcpy(d_z, &z, arrsizedouble, hipMemcpyHostToDevice);
  hipMemcpy(d_Xmin, &Xmin, sizedouble, hipMemcpyHostToDevice);
  hipMemcpy(d_Xmax, &Xmax, sizedouble, hipMemcpyHostToDevice);
  hipMemcpy(d_Ymin, &Ymin, sizedouble, hipMemcpyHostToDevice);
  hipMemcpy(d_Ymax, &Ymax, sizedouble, hipMemcpyHostToDevice);
  hipMemcpy(d_Zmin, &Zmin, sizedouble, hipMemcpyHostToDevice);
  hipMemcpy(d_Zmax, &Zmax, sizedouble, hipMemcpyHostToDevice);
  hipMemcpy(d_re, &re, sizedouble, hipMemcpyHostToDevice);
  hipMemcpy(d_DELTA, &DELTA, sizedouble, hipMemcpyHostToDevice);
  hipMemcpy(d_NUM, &NUM, sizeint, hipMemcpyHostToDevice);
  hipMemcpy(d_max_neighb, &MAX_NEIGHB, sizeint, hipMemcpyHostToDevice);
  hipMemcpy(d_sizeof_neighbours, &sizeof_neighbours, sizeint, hipMemcpyHostToDevice);

  // --------------- running the calcHash kernel ----------------------------------------

  calcHash<<<5,2>>>(d_x, d_y, d_z, d_particleHash, d_NUM, d_Xmax, d_Xmin, d_re, d_DELTA, d_Ymin, d_Ymax, d_Zmax, d_Zmin, d_particleid, d_tnc, d_ncx, d_ncy, d_ncz);

  // ---------------- sorting the particleHash array -----------------------------

  thrust::device_ptr<int> dev_Hash(d_particleHash);
  thrust::device_ptr<int> dev_id(d_particleid);
  thrust::sort_by_key(dev_Hash, dev_Hash + 10, dev_id); //need to generalise this 10

  hipMemcpy(particleHash, d_particleHash, arrsizeint, hipMemcpyDeviceToHost);
  hipMemcpy(particleid, d_particleid, arrsizeint, hipMemcpyDeviceToHost);

  

  // --------------------- finding cell start and cell end for each cell -----------------------------

  hipMemcpy(tnc, d_tnc, sizeint, hipMemcpyDeviceToHost);
  int cellarrsize = *tnc * sizeof(int);
  cellStart = (int *)malloc(cellarrsize);
  cellEnd = (int *)malloc(cellarrsize);
  hipMalloc((void **)&d_cellStart, cellarrsize); 
  hipMalloc((void **)&d_cellEnd, cellarrsize); 

  InitializeCellDetails<<<*tnc,1>>>(d_cellStart, d_cellEnd);
  findCellStart<<<5,2>>>(d_particleHash, d_cellStart, d_cellEnd, d_NUM);

  hipMemcpy(cellStart, d_cellStart, cellarrsize, hipMemcpyDeviceToHost);
  hipMemcpy(cellEnd, d_cellEnd, cellarrsize, hipMemcpyDeviceToHost);

  // -------------------------- Creating neighbour arrays for each particle ------------------------------

  //Template<<<5,2>>>(d_particleHash, d_particleid, d_cellStart, d_cellEnd, d_ncx, d_ncy, d_ncz, d_sizeof_neighbours, d_test);
  createNeighbourArraysCUDA<<<5,2>>>(d_neighb, d_cellStart, d_cellEnd, d_particleHash, d_particleid, d_ncx, d_ncy, d_ncz, d_max_neighb, d_test);
  test = (int *)malloc(arrsizeint);
  hipMemcpy(h_neighb, d_neighb, sizeneighb, hipMemcpyDeviceToHost);
  hipMemcpy(test, d_test, arrsizeint, hipMemcpyDeviceToHost);



  // ---------------------------- Populating neighb array ----------------------
  
       

  
  //neighb[10][50] = 5;
  /*
  
  for(int j=0; j<NUM; j++){
    for(int i=0; i<h_neighb[j*(MAX_NEIGHB + 1)]; i++){
      neighb[j+1][i+2] = 55;//h_neighb[j*(MAX_NEIGHB + 1) + i + 1];
    }
    neighb[j+1][1] = 55; //h_neighb[j*(MAX_NEIGHB + 1)];
  }
  */
  
  // ------------------ Debugging --------------------------

  cout<<endl<<" ParticleID - cellID "<<endl;
  for(int i=0; i<NUM; i++){
    cout<<particleid[i]<<" - "<<particleHash[i]<<endl;
  }

  cout<<endl<<"Neighbours new"<<endl;

  for(int i=0; i<NUM; i++){
    cout<<i<<" - "<<test[i]<<endl;
  }

  cout<<endl<<"Cell start and end new"<<endl;

  for(int i=0; i<*tnc; i++){
    cout<<i+1<<" - "<<cellStart[i]<<" : "<<cellEnd[i]<<endl;
  }

  cout<<"Neighbours according to the h_neighb array";

  for(int i=0; i<NUM; i++){
    int neighb_index = (MAX_NEIGHB + 1)*i;
    cout<<i<<" : "<<h_neighb[neighb_index]<<endl;
  }


  //after it all works, I am going to delete all unrequired host variables. 

  // -------------------------- Deallocating memory ---------------------------

  hipFree(d_particleHash);
  hipFree(d_particleid);
  hipFree(d_cellStart);
  hipFree(d_cellEnd);
  hipFree(d_x);
  hipFree(d_y);
  hipFree(d_z);
  hipFree(d_Xmin);
  hipFree(d_Xmax);
  hipFree(d_Ymin);
  hipFree(d_Ymax);
  hipFree(d_Zmin);
  hipFree(d_Zmax);
  hipFree(d_re);
  hipFree(d_NUM);
  hipFree(d_tnc);
  hipFree(d_ncx);
  hipFree(d_ncy);
  hipFree(d_ncz);
  hipFree(d_neighb);

  free(particleHash);
  free(particleid);
  free(cellStart);
  free(cellEnd);
  free(h_neighb);
  free(tnc);
}

void NEIGHBOUR_serial(){

  // ------------------PARAMETERS DEFENTION -------------------------------------
  int ncx = int((Xmax - Xmin) / (re + DELTA)) + 1;     // Number of cells in x direction
  int ncy = int((Ymax - Ymin) / (re + DELTA)) + 1;     // Number of cells in y direction
  int ncz = int((Zmax - Zmin) / (re + DELTA)) + 1;     // Number of cells in z direction

  int tnc = ncx*ncy*ncz;                 // Total number of cells   
  int m, k, kmax, Cnum;

  neighb = new int*[NUM+1];
  for(int i=0; i<NUM+1; i++){
    neighb[i] = new int[MAX_NEIGHB + 2];
  }


  int *Ista, *Iend, *nc, *icell, *jcell, *kcell;
  int *ip;                             // I is sorted number of ip[I] th paricle
  Ista = new int[tnc + 1]; //this points to the index of the first element in a cell in the array ip
  Iend = new int[tnc + 1]; //index of the last element in a cell in the array ip
  nc = new int[tnc + 1];
  icell = new int[NUM + 1];
  jcell = new int[NUM + 1];
  kcell = new int[NUM + 1];
  ip = new int[NUM + 1]; //this is the main array that we are looking for, it is sorted 
  // according to cell numbers and it contains particle indices 



  //----------------- ALLOCATING PRTICLES IN CELLS --------------------------


  for (k = 1; k <= tnc; k++) //cell loop 
  {
    Ista[k] = 1;
    Iend[k] = 0;
    nc[k] = 0;
  }
  cout<<"particle - cell - original"<<endl;
  for (k = 1; k <= NUM; k++) //particle loop
  {
    icell[k] = int((x[k-1] - Xmin) / (re + DELTA)) + 1;
    jcell[k] = int((y[k-1] - Ymin) / (re + DELTA)) + 1;
    kcell[k] = int((z[k-1] - Zmin) / (re + DELTA)) + 1;

    Cnum = icell[k] + (jcell[k] - 1)*ncx + (kcell[k] - 1)*ncx*ncy;     // Cell number in which particle k located
    cout<<k<<" "<<Cnum<<endl;

    nc[Cnum]++;                       // Number of particle in cell Cnum
    Iend[Cnum]++;                   // Number of particle in cell Cnum 

    for (m = Iend[tnc]; m >= Iend[Cnum]; m--)
    {
      if (m>0) ip[m + 1] = ip[m];
    } //this block is there to create space at the end as and when new particles are added

    for (m = Cnum + 1; m <= tnc; m++)
    {
      Ista[m]++;
      Iend[m]++;
    }

    ip[Iend[Cnum]] = k;
  }

  cout<<endl<<"cell start and cell end original"<<endl;

  for(int i=1; i<=tnc; i++){
    cout<<i<<" - "<<Ista[i]<<" : "<<Iend[i]<<endl;
  }


  //--------------- FINDIND NEIGHBORS ----------------------------------
  int JJ, J;
  for (int I = 1; I <= NUM; I++)
  {
    k = 2;
    int row, colu, elev, m1, m2, m3, m4, m5, m6;
    if (icell[I] == 1)m1 = 0; else m1 = -1;
    if (icell[I] == ncx)m2 = 0; else m2 = +1;
    if (jcell[I] == 1)m3 = 0; else m3 = -1;
    if (jcell[I] == ncy)m4 = 0; else m4 = +1;
    if (kcell[I] == 1)m5 = 0; else m5 = -1;
    if (kcell[I] == ncz)m6 = 0; else m6 = +1;

    for (row = m1; row <= m2; row++) //could be -1 to 1 , the triple loop is basically there to find all the 9 cells around that particle, including the one in which it itself is
    {
      for (colu = m3; colu <= m4; colu++) 
      {
        for (elev = m5; elev <= m6; elev++)
        {

          Cnum = icell[I] + row + (jcell[I] - 1 + colu)*ncx + (kcell[I] - 1 + elev)*ncx*ncy;

          for (JJ = Ista[Cnum]; JJ <= Iend[Cnum]; JJ++)
          {
            J = ip[JJ]; //J is tha ACTUAL particle index 
            neighb[I][k] = J;
            k++;
          }
        }
      }
    }
    kmax = k - 2;
    neighb[I][1] = kmax; //this is the total number of neighbours, which is stored at the beginning 
    //if( neighb[I][1]>1098 ||neighb[I][1]*0!=0) printf("ERROR, the neighbors of particles %d is %d", I, neighb[I][1]);
  }
  //--------------------Clearing dynamic arrays ----------------------------

  delete[]Ista;
  delete[]Iend;
  delete[]nc;
  delete[]icell;
  delete[]jcell;
  delete[]kcell;
  delete[]ip;
  Ista = NULL; Iend = NULL; nc = NULL; icell = NULL; jcell = NULL; kcell = NULL, ip = NULL;
}


// -------------------- host sub-routine for neighbour calculation --------------------------

int main(){

	NEIGHBOUR_cuda();
  NEIGHBOUR_serial();
  cout<<endl<<"original neighbours";
  for(int i=1; i<=NUM; i++){
  cout<<endl<<neighb[i][1];
  }


	return 0;
} 