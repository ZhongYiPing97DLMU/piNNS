#include "hip/hip_runtime.h"
/*
Copyright (C) 2018-2020 Deep Tavker (tavkerdeep@gmail.com)
Copyright (C) 2020 André Luiz Vieira-e-Silva (albvs@cin.ufpe.br)

This program is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.
This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.
You should have received a copy of the GNU General Public License
along with this program.  If not, see <https://www.gnu.org/licenses/>.
*/


#include <algorithm>
#include <stdio.h>
#include <cmath>
#include <fstream>
#include <iostream>
#include <time.h>
#include <vector>

#include <thrust/sort.h>
#include <thrust/device_ptr.h>
#include "neighbour_parallel.h"
#include "common.h"
#include ""


// ----------------- CUDA KERNELS -------------------------

__global__ void calcHash(double *d_x, double *d_y, int *d_particleHash, \
	int *d_TP, double *d_Xmax, double *d_Xmin, double *d_re, double *d_DELTA, double *d_Ymin, \
	double *d_Ymax, int *d_particleid, int *d_tnc, int *ncx, \
	int *ncy) {

	int k =  threadIdx.x + blockIdx.x * blockDim.x;
	if (k < *d_TP) {


		*ncx = int((*d_Xmax - *d_Xmin) / (*d_re + *d_DELTA)) + 1;     // Number of cells in x direction
		*ncy = int((*d_Ymax - *d_Ymin) / (*d_re + *d_DELTA)) + 1;     // Number of cells in y direction
		*d_tnc = *ncx * *ncy;


		int *icell, *jcell, *cellNum;

		int sizeint = sizeof(int);
		icell = (int *)malloc(sizeint);
		jcell = (int *)malloc(sizeint);
		cellNum = (int *)malloc(sizeint);

		*icell = int((d_x[k + 1] - *d_Xmin) / (*d_re + *d_DELTA)) + 1;
		*jcell = int((d_y[k + 1] - *d_Ymin) / (*d_re + *d_DELTA)) + 1;

		*cellNum = *icell + (*jcell - 1)* *ncx;

		d_particleHash[k] = *cellNum;
		d_particleid[k] = k + 1;
		//particlehash and particleId have indices starting from 0 which corresponds to the index 1 in the coordinate array (which is x, y, and z)
		//bsically x and y will have sizes TP+1
		free(icell);
		free(jcell);
		free(cellNum);
	}

}

__global__ void findCellStart(int *particleHash, int *cellStart, int *cellEnd, int *TP) {

	int k =  threadIdx.x + blockIdx.x * blockDim.x; // here index value corresponds to index of the array particleHash
	//cellNum will assume values starting from 1 but the corresponding cellstart and cellend will start from first index as 0
	if (k < *TP) {
		if (particleHash[k] != particleHash[k + 1] && k != *TP - 1) {
			cellEnd[particleHash[k] - 1] = k;
			cellStart[particleHash[k + 1] - 1] = k + 1;
		}
		if (k == *TP - 1) {
			cellEnd[particleHash[k] - 1] = k;
		}
	}

	free(&k);
}

__global__ void createNeighbourArraysCUDA(int *d_neighb, int *cellStart, int *cellEnd, int *particleHash, int *particleid, int *ncx, int *ncy, int *d_max_neighb, int *d_TP, double *d_re, double *d_DELTA, double *d_x, double *d_y) {


	int index = threadIdx.x + blockIdx.x * blockDim.x;

	if (index < *d_TP) {
		int pid, icell, jcell, cellNum, neighb_index, Cnum, J, curr_neighb_num, m1, m2, m3, m4;

		double R;

		cellNum = particleHash[index];
		pid = particleid[index];

		neighb_index = (pid - 1) * (*d_max_neighb + 1) ;

		jcell = ((cellNum - 1) / *ncx) + 1;
		icell = cellNum - *ncx * (jcell - 1);

		curr_neighb_num = 0;

		if (icell == 1) m1 = 0; else m1 = -1;
		if (icell == *ncx) m2 = 0; else m2 = +1;
		if (jcell == 1) m3 = 0; else m3 = -1;
		if (jcell == *ncy) m4 = 0; else m4 = +1;

		for (int row = m1; row <= m2; row++)
		{
			for (int colu = m3; colu <= m4; colu++)
			{

				Cnum = icell + row + (jcell - 1 + colu)* *ncx;

				if (cellEnd[Cnum - 1] != -1) {

					for (int JJ = cellStart[Cnum - 1]; JJ <= cellEnd[Cnum - 1]; JJ++)
					{
						J = particleid[JJ];
						R = sqrt(pow(d_x[J] - d_x[pid], 2.0) + pow(d_y[J] - d_y[pid], 2.0));
						if (R <= *d_re + *d_DELTA) {
							curr_neighb_num = curr_neighb_num + 1;
							d_neighb[neighb_index + curr_neighb_num] = J; //here the index is shifted by one unit to conform to the original MPS convention
						}
					}
				}

			}
		}


		d_neighb[neighb_index] = curr_neighb_num;

	}
}

__global__ void createNeighbourArraysCUDAgpu(int offset, int *d_neighb, int *cellStart, int *cellEnd, int *particleHash, int *particleid, int *ncx, int *ncy, int *d_max_neighb, int *d_TP, double *d_re, double *d_DELTA, double *d_x, double *d_y) {


	int index = threadIdx.x + blockIdx.x * blockDim.x + offset;

	if (index < *d_TP) {
		int pid, icell, jcell, cellNum, neighb_index, Cnum, J, curr_neighb_num, m1, m2, m3, m4;

		double R;

		cellNum = particleHash[index];
		pid = particleid[index];

		neighb_index = pid == 1? 1 : (pid - 1) * (*d_max_neighb /*+ 1*/) + 1;
		neighb_index = neighb_index + (*d_max_neighb);

		jcell = ((cellNum - 1) / *ncx) + 1;
		icell = cellNum - *ncx * (jcell - 1);

		curr_neighb_num = 0;

		if (icell == 1) m1 = 0; else m1 = -1;
		if (icell == *ncx) m2 = 0; else m2 = +1;
		if (jcell == 1) m3 = 0; else m3 = -1;
		if (jcell == *ncy) m4 = 0; else m4 = +1;

		for (int row = m1; row <= m2; row++)
		{
			for (int colu = m3; colu <= m4; colu++)
			{

				Cnum = icell + row + (jcell - 1 + colu)* *ncx;
				
				if (cellEnd[Cnum - 1] != -1) {

					for (int JJ = cellStart[Cnum - 1]; JJ <= cellEnd[Cnum - 1]; JJ++)
					{
						J = particleid[JJ];
						R = sqrt(pow(d_x[J] - d_x[pid], 2.0) + pow(d_y[J] - d_y[pid], 2.0));
						if (R <= *d_re + *d_DELTA) {
							curr_neighb_num = curr_neighb_num + 1;
							d_neighb[neighb_index + curr_neighb_num] = J; //here the index is shifted by one unit to conform to the original MPS convention
						}
					}
				}

			}
		}


		//d_neighb[neighb_index] = curr_neighb_num;
		d_neighb[neighb_index] = curr_neighb_num+1;

	}
}

__global__ void InitializeCellDetails(int *cellStart, int *cellEnd, int *d_tnc) {
	int index =  threadIdx.x + blockIdx.x * blockDim.x;
	if (index < *d_tnc) {
		cellStart[index] = 0; cellEnd[index] = -1;
	}
	free(&index);
}




// ------------------------- Host sub-sub-routine for neighbour computation ------------------------ 




void neighbour_cuda_2d(int TP, double *x, double *y, double DELTA, double re, int ** neighb, double Xmin, double Xmax, double Ymin, double Ymax) {

	int MAX_NEIGHB = 300, THREADS_PER_BLOCK = 256;

	// ------------------ variable declarations and initializations ------------------------------

	int *d_cellEnd, *d_cellStart, *d_TP, *d_tnc, *tnc, *d_ncx, *d_ncy, *d_max_neighb;
	int *d_particleHash, *d_particleid, *d_neighb, *h_neighb, *d_sizeof_neighbours;
	double *d_x, *d_y, *d_Xmax, *d_Xmin, *d_Ymax, *d_Ymin, *d_re, *d_DELTA;

	int arrsizeint = TP * sizeof(int);
	int sizeint = sizeof(int);
	int arrsizedouble = (TP + 1) * sizeof(double);
	int sizedouble = sizeof(double);
	int sizeneighb = TP * (MAX_NEIGHB + 1) * sizeof(int);
	int sizeof_neighbours = (MAX_NEIGHB + 1) * sizeof(int);

	tnc = (int *)malloc(sizeint);
	h_neighb = (int *)malloc(sizeneighb);



	hipMalloc((void **)&d_particleHash, arrsizeint);
	hipMalloc((void **)&d_particleid, arrsizeint);
	hipMalloc((void **)&d_x, arrsizedouble);
	hipMalloc((void **)&d_y, arrsizedouble);
	hipMalloc((void **)&d_Xmin, sizedouble);
	hipMalloc((void **)&d_Xmax, sizedouble);
	hipMalloc((void **)&d_Ymin, sizedouble);
	hipMalloc((void **)&d_Ymax, sizedouble);
	hipMalloc((void **)&d_re, sizedouble);
	hipMalloc((void **)&d_DELTA, sizedouble);
	hipMalloc((void **)&d_TP, sizeint);
	hipMalloc((void **)&d_tnc, sizeint);
	hipMalloc((void **)&d_ncx, sizeint);
	hipMalloc((void **)&d_ncy, sizeint);
	hipMalloc((void **)&d_neighb, sizeneighb);
	hipMalloc((void **)&d_max_neighb, sizeint);
	hipMalloc((void **)&d_sizeof_neighbours, sizeof_neighbours);

	hipMemcpy(d_x, x, arrsizedouble, hipMemcpyHostToDevice);
	hipMemcpy(d_y, y, arrsizedouble, hipMemcpyHostToDevice);
	hipMemcpy(d_Xmin, &Xmin, sizedouble, hipMemcpyHostToDevice);
	hipMemcpy(d_Xmax, &Xmax, sizedouble, hipMemcpyHostToDevice);
	hipMemcpy(d_Ymin, &Ymin, sizedouble, hipMemcpyHostToDevice);
	hipMemcpy(d_Ymax, &Ymax, sizedouble, hipMemcpyHostToDevice);
	hipMemcpy(d_re, &re, sizedouble, hipMemcpyHostToDevice);
	hipMemcpy(d_DELTA, &DELTA, sizedouble, hipMemcpyHostToDevice);
	hipMemcpy(d_TP, &TP, sizeint, hipMemcpyHostToDevice);
	hipMemcpy(d_max_neighb, &MAX_NEIGHB, sizeint, hipMemcpyHostToDevice);
	hipMemcpy(d_sizeof_neighbours, &sizeof_neighbours, sizeint, hipMemcpyHostToDevice);




	// --------------- running the calcHash kernel ----------------------------------------

	calcHash << <TP / THREADS_PER_BLOCK + 1, THREADS_PER_BLOCK >> > (d_x, d_y, d_particleHash, d_TP, d_Xmax, d_Xmin, d_re, d_DELTA, d_Ymin, d_Ymax, d_particleid, d_tnc, d_ncx, d_ncy);

	// ---------------- sorting the particleHash array -----------------------------

	thrust::device_ptr<int> dev_Hash(d_particleHash);
	thrust::device_ptr<int> dev_id(d_particleid);
	thrust::sort_by_key(dev_Hash, dev_Hash + TP, dev_id); //need to generalise this 10



	// --------------------- finding cell start and cell end for each cell -----------------------------

	hipMemcpy(tnc, d_tnc, sizeint, hipMemcpyDeviceToHost);
	int cellarrsize = *tnc * sizeof(int);

	hipMalloc((void **)&d_cellStart, cellarrsize);
	hipMalloc((void **)&d_cellEnd, cellarrsize);


	InitializeCellDetails << <*tnc / THREADS_PER_BLOCK + 1, THREADS_PER_BLOCK >> > (d_cellStart, d_cellEnd, d_tnc);
	findCellStart << <TP / THREADS_PER_BLOCK + 1, THREADS_PER_BLOCK >> > (d_particleHash, d_cellStart, d_cellEnd, d_TP);


	// -------------------------- Creating neighbour arrays for each particle ------------------------------


	createNeighbourArraysCUDA << <TP / THREADS_PER_BLOCK + 1, THREADS_PER_BLOCK >> > (d_neighb, d_cellStart, d_cellEnd, d_particleHash, d_particleid, d_ncx, d_ncy, d_max_neighb, d_TP, d_re, d_DELTA, d_x, d_y);



	hipMemcpy(h_neighb, d_neighb, sizeneighb, hipMemcpyDeviceToHost);



	// ---------------------------- Populating neighb array ----------------------


	for (int j = 0; j < TP; j++) {
		for (int i = 0; i < h_neighb[j*(MAX_NEIGHB + 1)]; i++) {
			neighb[j + 1][i + 2] = h_neighb[j*(MAX_NEIGHB + 1) + i + 1];
		}
		neighb[j + 1][1] = h_neighb[j*(MAX_NEIGHB + 1)] + 1;
	}


	// -------------------------- Deallocating memory ---------------------------

	hipFree(d_particleHash);
	hipFree(d_particleid);
	hipFree(d_cellStart);
	hipFree(d_cellEnd);
	hipFree(d_x);
	hipFree(d_y);
	hipFree(d_Xmin);
	hipFree(d_Xmax);
	hipFree(d_Ymin);
	hipFree(d_Ymax);
	hipFree(d_re);
	hipFree(d_TP);
	hipFree(d_tnc);
	hipFree(d_ncx);
	hipFree(d_ncy);
	hipFree(d_neighb);
	hipFree(d_max_neighb);
	hipFree(d_sizeof_neighbours);

	free(h_neighb);
	free(tnc);
}

void neighbour_cuda_2d_gpu(int TP, double *d_x, double *d_y, double DELTA, double re, int *d_neighb, double Xmin, double Xmax, double Ymin, double Ymax) {

	int MAX_NEIGHB = 300, THREADS_PER_BLOCK = 256;


	// ------------------ variable declarations and initializations ------------------------------

	int *d_cellEnd, *d_cellStart, *d_TP, *d_tnc, *tnc, *d_ncx, *d_ncy, *d_max_neighb;
	int *d_particleHash, *d_particleid, *h_neighb, *d_sizeof_neighbours;
	double *d_Xmax, *d_Xmin, *d_Ymax, *d_Ymin, *d_re, *d_DELTA;

	int arrsizeint = TP * sizeof(int);
	int sizeint = sizeof(int);
	int sizedouble = sizeof(double);
	int sizeneighb = (TP+1) * (MAX_NEIGHB + 1) * sizeof(int);
	int sizeof_neighbours = (MAX_NEIGHB + 1) * sizeof(int);

	tnc = (int *)malloc(sizeint);
	h_neighb = (int *)malloc(sizeneighb);


	hipMalloc((void **)&d_particleHash, arrsizeint);
	hipMalloc((void **)&d_particleid, arrsizeint);
	hipMalloc((void **)&d_Xmin, sizedouble);
	hipMalloc((void **)&d_Xmax, sizedouble);
	hipMalloc((void **)&d_Ymin, sizedouble);
	hipMalloc((void **)&d_Ymax, sizedouble);
	hipMalloc((void **)&d_re, sizedouble);
	hipMalloc((void **)&d_DELTA, sizedouble);
	hipMalloc((void **)&d_TP, sizeint);
	hipMalloc((void **)&d_tnc, sizeint);
	hipMalloc((void **)&d_ncx, sizeint);
	hipMalloc((void **)&d_ncy, sizeint);
	hipMalloc((void **)&d_max_neighb, sizeint);
	hipMalloc((void **)&d_sizeof_neighbours, sizeof_neighbours);

	hipMemcpy(d_Xmin, &Xmin, sizedouble, hipMemcpyHostToDevice);
	hipMemcpy(d_Xmax, &Xmax, sizedouble, hipMemcpyHostToDevice);
	hipMemcpy(d_Ymin, &Ymin, sizedouble, hipMemcpyHostToDevice);
	hipMemcpy(d_Ymax, &Ymax, sizedouble, hipMemcpyHostToDevice);
	hipMemcpy(d_re, &re, sizedouble, hipMemcpyHostToDevice);
	hipMemcpy(d_DELTA, &DELTA, sizedouble, hipMemcpyHostToDevice);
	hipMemcpy(d_TP, &TP, sizeint, hipMemcpyHostToDevice);
	hipMemcpy(d_max_neighb, &MAX_NEIGHB, sizeint, hipMemcpyHostToDevice);
	hipMemcpy(d_sizeof_neighbours, &sizeof_neighbours, sizeint, hipMemcpyHostToDevice);




	// --------------- running the calcHash kernel ----------------------------------------

	calcHash << <TP / THREADS_PER_BLOCK + 1, THREADS_PER_BLOCK >> > (d_x, d_y, d_particleHash, d_TP, d_Xmax, d_Xmin, d_re, d_DELTA, d_Ymin, d_Ymax, d_particleid, d_tnc, d_ncx, d_ncy);

	// ---------------- sorting the particleHash array -----------------------------

	thrust::device_ptr<int> dev_Hash(d_particleHash);
	thrust::device_ptr<int> dev_id(d_particleid);
	thrust::sort_by_key(dev_Hash, dev_Hash + TP, dev_id); //need to generalise this 10



	// --------------------- finding cell start and cell end for each cell -----------------------------

	hipMemcpy(tnc, d_tnc, sizeint, hipMemcpyDeviceToHost);
	int cellarrsize = *tnc * sizeof(int);

	hipMalloc((void **)&d_cellStart, cellarrsize);
	hipMalloc((void **)&d_cellEnd, cellarrsize);


	InitializeCellDetails << <*tnc / THREADS_PER_BLOCK + 1, THREADS_PER_BLOCK >> > (d_cellStart, d_cellEnd, d_tnc);
	findCellStart << <TP / THREADS_PER_BLOCK + 1, THREADS_PER_BLOCK >> > (d_particleHash, d_cellStart, d_cellEnd, d_TP);


	// -------------------------- Creating neighbour arrays for each particle ------------------------------

	hipMemset(d_neighb, 0, sizeneighb);

	createNeighbourArraysCUDAgpu << <TP / THREADS_PER_BLOCK + 1, THREADS_PER_BLOCK >> > (0, d_neighb, d_cellStart, d_cellEnd, d_particleHash, d_particleid, d_ncx, d_ncy, d_max_neighb, d_TP, d_re, d_DELTA, d_x, d_y);

	// -------------------------- Deallocating memory ---------------------------

	hipFree(d_particleHash);
	hipFree(d_particleid);
	hipFree(d_cellStart);
	hipFree(d_cellEnd);
	hipFree(d_Xmin);
	hipFree(d_Xmax);
	hipFree(d_Ymin);
	hipFree(d_Ymax);
	hipFree(d_re);
	hipFree(d_TP);
	hipFree(d_tnc);
	hipFree(d_ncx);
	hipFree(d_ncy);
	hipFree(d_max_neighb);
	hipFree(d_sizeof_neighbours);

	free(h_neighb);
	free(tnc);
}
