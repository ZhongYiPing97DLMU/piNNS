#include "hip/hip_runtime.h"
#include <algorithm>
#include <stdio.h>
#include <math.h>
#include <fstream>
#include <iostream>
#include <time.h>
#include <vector>
#include <thrust/sort.h>

using namespace std;

double x[10]={1,1.001,3,4,5,6,7,8,9,0};
double y[10]={1,1.001,3,4,5,6,7,8,9,0};
double z[10]={1,1.001,3,4,5,6,7,8,9,0};
double Xmax=9, Xmin=0;
double Ymax=9, Ymin=0;
double Zmax=9, Zmin=0;
double re=5, DELTA=0;
int NUM=10;
int **neighb;


//declarations of global variables such as x, y, z, re, DELTA, xmax, etc/ 

//CUDA kernel for calcHash
__global__ void calcHash(double *d_x, double *d_y, double *d_z, int *d_particleHash,\
	int *d_NUM, double *d_Xmax, double *d_Xmin, double *d_re, double *d_DELTA, double *d_Ymin, \
  double *d_Ymax, double *d_Zmax, double *d_Zmin, int *d_particleid, int *d_tnc, int *ncx, int *ncy,\
  int *ncz){

  *ncx = int((*d_Xmax - *d_Xmin) / (*d_re + *d_DELTA)) + 1;     // Number of cells in x direction
  *ncy = int((*d_Ymax - *d_Ymin) / (*d_re + *d_DELTA)) + 1;     // Number of cells in y direction
  *ncz = int((*d_Zmax - *d_Zmin) / (*d_re + *d_DELTA)) + 1;     // Number of cells in z direction
  *d_tnc = *ncx * *ncy * *ncz;

  int k = threadIdx.x + blockIdx.x * blockDim.x;
  int *icell, *jcell, *kcell;

  int cellarrsize =  *d_NUM * sizeof(int);
  icell = (int *)malloc(cellarrsize);
  jcell = (int *)malloc(cellarrsize);
  kcell = (int *)malloc(cellarrsize);
  
  icell[k] = int((d_x[k] - *d_Xmin) / (*d_re + *d_DELTA)) + 1;
  jcell[k] = int((d_y[k] - *d_Ymin) / (*d_re + *d_DELTA)) + 1;
  kcell[k] = int((d_z[k] - *d_Zmin) / (*d_re + *d_DELTA)) + 1;

  int cellNum = icell[k] + (jcell[k] - 1)* *ncx + (kcell[k] - 1)* *ncx * *ncy;

  d_particleHash[k] = cellNum;
  d_particleid[k] = k;

}

__global__ void findCellStart(int *particleHash, int *cellStart, int *cellEnd){

  int k = threadIdx.x + blockIdx.x * blockDim.x; // here index value is equal to the cell number which starts with 1 
  if (particleHash[k] != particleHash[k+1] and k!=9){
    cellEnd[particleHash[k] - 1] = k;
    cellStart[particleHash[k+1] - 1] = k+1;
  }
  if(k == 9){
    cellEnd[particleHash[k] - 1] = k;
  }
              
}

//CUDA kernel for neighb[][] creation

__global__ void createNeighbourArrays(int **neighb, int *cellStart, int *cellEnd, int *particleHash, int *particleid, int *ncx, int *ncy, int *ncz){

  int index = threadIdx.x + blockIdx.x * blockDim.x; 
  //neighb[particleid[k]][2, 3, 4, 5, 6, 7....] = ids of other particles in that cell 
  int pid, icell, jcell, kcell, cellNum;

  cellNum = particleHash[index]; pid = particleid[index];
  kcell = (cellNum - 1)/((*ncx) * (*ncy)) + 1;
  jcell = ((cellNum - 1) - ((kcell - 1)* (*ncx) * (*ncy)))/ *ncx + 1;
  icell = cellNum - 1 - *ncx * (jcell - 1) - (*ncx * *ncy)*(kcell - 1) + 1;

  int k = 2;
  int Cnum, J;

  int row, colu, elev, m1, m2, m3, m4, m5, m6;
  if (icell == 1)m1 = 0; else m1 = -1;
  if (icell == *ncx)m2 = 0; else m2 = +1;
  if (jcell == 1)m3 = 0; else m3 = -1;
  if (jcell == *ncy)m4 = 0; else m4 = +1;
  if (kcell == 1)m5 = 0; else m5 = -1;
  if (kcell == *ncz)m6 = 0; else m6 = +1;

  for (row = m1; row <= m2; row++)
  {
    for (colu = m3; colu <= m4; colu++) 
    {
      for (elev = m5; elev <= m6; elev++)
      {

        Cnum = icell + row + (jcell - 1 + colu)* *ncx + (kcell - 1 + elev)* *ncx* *ncy;

        for (int JJ = cellStart[Cnum]; JJ <= cellEnd[Cnum]; JJ++)
        {
          J = particleid[JJ]; //J is tha ACTUAL particle index 
          neighb[pid][k] = J;
          k++;
        }
      }
    }
  }
  int kmax = k - 2;
  neighb[pid][1] = kmax;

}

__global__ void InitializeCellDetails(int *cellStart, int *cellEnd){
  int index = threadIdx.x + blockIdx.x * blockDim.x; 
  cellStart[index] = 0; cellEnd[index] = 0;
}



void neighbour_cuda(){

  // ------------------ variable declarations and initializations ------------------------------

  int *d_cellEnd, *d_cellStart, *d_NUM, *cellStart, *cellEnd, *d_tnc, *tnc, *d_ncx, *d_ncy, *d_ncz;
  int *particleHash, *d_particleHash, *d_particleid, *particleid, **d_neighb;
  double *d_x, *d_y, *d_z, *d_Xmax, *d_Xmin, *d_Ymax, *d_Ymin, *d_Zmax, *d_Zmin, *d_re, *d_DELTA;

  int arrsizeint = NUM * sizeof(int);
  int sizeint = sizeof(int);
  int arrsizedouble = NUM * sizeof(double);
  int sizedouble = sizeof(double);
  int sizeneighb = NUM * sizeof(int*);
  
  size_t *d_pitch, *pitch;

  particleHash = (int *)malloc(arrsizeint);
  particleid = (int *)malloc(arrsizeint);
  tnc = (int *)malloc(sizeint);
  
  

  hipMalloc((void **)&d_particleHash, arrsizeint);
  hipMalloc((void **)&d_particleid, arrsizeint); 
  
  hipMalloc((void **)&d_x, arrsizedouble);
  hipMalloc((void **)&d_y, arrsizedouble);
  hipMalloc((void **)&d_z, arrsizedouble);
  hipMalloc((void **)&d_Xmin, sizedouble);
  hipMalloc((void **)&d_Xmax, sizedouble);
  hipMalloc((void **)&d_Ymin, sizedouble);
  hipMalloc((void **)&d_Ymax, sizedouble);
  hipMalloc((void **)&d_Zmin, sizedouble);
  hipMalloc((void **)&d_Zmax, sizedouble);
  hipMalloc((void **)&d_re, sizedouble);
  hipMalloc((void **)&d_DELTA, sizedouble);
  hipMalloc((void **)&d_NUM, sizeint);
  hipMalloc((void **)&d_tnc, sizeint);
  hipMalloc((void **)&d_ncx, sizeint);
  hipMalloc((void **)&d_ncy, sizeint);
  hipMalloc((void **)&d_ncz, sizeint);

  hipMallocPitch(d_neighb, d_pitch, 1500*sizeof(int), NUM);

  hipMemcpy(d_x, &x, arrsizedouble, hipMemcpyHostToDevice);
  hipMemcpy(d_y, &y, arrsizedouble, hipMemcpyHostToDevice);
  hipMemcpy(d_z, &z, arrsizedouble, hipMemcpyHostToDevice);
  hipMemcpy(d_Xmin, &Xmin, sizedouble, hipMemcpyHostToDevice);
  hipMemcpy(d_Xmax, &Xmax, sizedouble, hipMemcpyHostToDevice);
  hipMemcpy(d_Ymin, &Ymin, sizedouble, hipMemcpyHostToDevice);
  hipMemcpy(d_Ymax, &Ymax, sizedouble, hipMemcpyHostToDevice);
  hipMemcpy(d_Zmin, &Zmin, sizedouble, hipMemcpyHostToDevice);
  hipMemcpy(d_Zmax, &Zmax, sizedouble, hipMemcpyHostToDevice);
  hipMemcpy(d_re, &re, sizedouble, hipMemcpyHostToDevice);
  hipMemcpy(d_DELTA, &DELTA, sizedouble, hipMemcpyHostToDevice);
  hipMemcpy(d_NUM, &NUM, sizeint, hipMemcpyHostToDevice);

  // --------------- running the calcHash kernel ----------------------------------------

  calcHash<<<5,2>>>(d_x, d_y, d_z, d_particleHash, d_NUM, d_Xmax, d_Xmin, d_re, d_DELTA, d_Ymin, d_Ymax, d_Zmax, d_Zmin, d_particleid, d_tnc, d_ncx, d_ncy, d_ncz);

  // ---------------- sorting the particleHash array -----------------------------

  thrust::device_ptr<int> dev_Hash(d_particleHash);
  thrust::device_ptr<int> dev_id(d_particleid);
  //thrust::device_ptr<int> dev_NUM(d_NUM);
  thrust::sort_by_key(dev_Hash, dev_Hash + 10, dev_id); //need to generalise this 10

  hipMemcpy(particleHash, d_particleHash, arrsizeint, hipMemcpyDeviceToHost);
  hipMemcpy(particleid, d_particleid, arrsizeint, hipMemcpyDeviceToHost);


  for(int k=0; k<10; k++){
  cout<<particleHash[k]<<"  "<<particleid[k]<<endl;
  }

  // --------------------- finding cell start and cell end for each cell -----------------------------

  hipMemcpy(tnc, d_tnc, sizeint, hipMemcpyDeviceToHost);
  int cellarrsize = *tnc * sizeof(int);
  cellStart = (int *)malloc(cellarrsize);
  cellEnd = (int *)malloc(cellarrsize);
  hipMalloc((void **)&d_cellStart, cellarrsize); //need to initialise this with all zeros
  hipMalloc((void **)&d_cellEnd, cellarrsize); //need to initialise this with all zeros 

  InitializeCellDetails<<<*tnc,1>>>(d_cellStart, d_cellEnd);
  findCellStart<<<5,2>>>(d_particleHash, d_cellStart, d_cellEnd);

  hipMemcpy(cellStart, d_cellStart, cellarrsize, hipMemcpyDeviceToHost);
  hipMemcpy(cellEnd, d_cellEnd, cellarrsize, hipMemcpyDeviceToHost);

  for(int i=0; i< *tnc; i++){
    cout<<cellStart[i]<<" "<<cellEnd[i]<<endl;
  }




  // -------------------------- Creating neighbour arrays for each particle ------------------------------

  createNeighbourArrays<<<5,2>>>(d_neighb, d_cellStart, d_cellEnd, d_particleHash, d_particleid, d_ncx, d_ncy, d_ncz);

  neighb = (int **)malloc(sizeof(int *)*NUM);
  for(int i=0; i<NUM; i++){
    neighb[i] = (int *)malloc(1500*sizeof(int));
  }

  hipMemcpy2D(neighb, *pitch, d_neighb, *d_pitch, 1500*sizeof(int), NUM, hipMemcpyDeviceToHost);

  cout<<neighb[1][1];


  hipFree(d_particleHash);
  hipFree(d_particleid);
  hipFree(d_cellStart);
  hipFree(d_cellEnd);
  hipFree(d_x);
  hipFree(d_y);
  hipFree(d_z);
  hipFree(d_Xmin);
  hipFree(d_Xmax);
  hipFree(d_Ymin);
  hipFree(d_Ymax);
  hipFree(d_Zmin);
  hipFree(d_Zmax);
  hipFree(d_re);
  hipFree(d_NUM);
  hipFree(d_tnc);
  hipFree(d_ncx);
  hipFree(d_ncy);
  hipFree(d_ncz);
  hipFree(d_neighb);

  free(particleHash);
  free(particleid);
  free(cellStart);
  free(cellEnd);
  //free(x);
  //free(y);
  //free(z);
  //free(Xmin);
  //free(Xmax);
  //free(Ymin);
  //free(Ymax);
  //free(Zmin);
  //free(Zmax);
  //free(re);
  //free(NUM);
  free(tnc);
}

int main(){

	neighbour_cuda();
	return 0;
}